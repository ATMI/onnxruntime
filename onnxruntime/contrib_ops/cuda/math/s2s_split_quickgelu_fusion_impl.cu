#include "hip/hip_runtime.h"
// Copyright (c) Microsoft Corporation. All rights reserved.
// Licensed under the MIT License.

#include "core/providers/cuda/cu_inc/common.cuh"
#include "contrib_ops/cuda/math/s2s_split_quickgelu_fusion_impl.h"

using namespace onnxruntime::cuda;

namespace onnxruntime {
namespace contrib {
namespace cuda {

namespace {

// constexpr int kElementsPerThread = GridDim::maxElementsPerThread;
// #ifdef USE_ROCM
// constexpr int kThreadsPerBlock = 512;
// #else
// constexpr int kThreadsPerBlock = GridDim::maxThreadsPerBlock;
// #endif

}  // namespace

// Need to use SplitSameSplitDimImpl (the other one works for different split sizes)

template <typename T>
__global__ void S2SModelSplitQuickGeluKernel(const int num_outputs, const T* input, T* output) {
  uint dim = 2;
  uint max_len = 16;
  float alpha = 1.702f;
  uint max_dim = 4;
  T one = static_cast<T>(1.f);
  T zero = static_cast<T>(0.f);
  for (uint i = 0; i < max_dim; i++){
    for (uint j = 0; j < dim; j++){
      T v = input[dim + i*max_dim+j] * static_cast<T>(alpha);
      T sigmoid = v >= zero ? one / (one + _Exp(-v)) : one - one / (one + _Exp(v));
      T quickgelu_out = input[dim + i*max_dim+j] * sigmoid;
      output[i*max_dim/2+j] = input[i*max_dim+j] * quickgelu_out;
    }
  }
  // for (uint i = 0; i < max_len / 2; i++) {
  //   T v = input[dim + i] * static_cast<T>(alpha);
  //   T one = static_cast<T>(1.f);
  //   T zero = static_cast<T>(0.f);
  //   T sigmoid = v >= zero ? one / (one + _Exp(-v)) : one - one / (one + _Exp(v));
  //   output[i] = input[i] * sigmoid;
  // }
}

// template <typename T>
// __global__ void VectorizedS2SModelSplitQuickGeluKernel(int64_t axis, const T* X, T* Y) {
//   const auto kElementsPerBlock = kElementsPerThread * blockDim.x;
//   const auto bias_idx = kElementsPerBlock * blockIdx.y + kElementsPerThread * threadIdx.x;
//   if (bias_idx >= bias_size) {
//     return;
//   }

//   const auto input_idx = bias_size * blockIdx.x + kElementsPerBlock * blockIdx.y + kElementsPerThread * threadIdx.x;

//   using LoadT = aligned_vector<T, kElementsPerThread>;

//   T reg_X[kElementsPerThread];
//   T reg_Y[kElementsPerThread];

// void LaunchS2SModelSplitQuickGeluKernel(hipStream_t stream, const size_t element_size, const int block_size_including_axis_dim,
//                                         const int block_size_inside_axis_dim, const int64_t split_size, const int num_outputs,
//                                         const void* input_data, OutputDataArray output_data, const size_t input_size) {

template <typename T>
void LaunchS2SModelSplitQuickGeluKernel(hipStream_t stream, int num_outputs, const T* input_data, T* output_data) {
  // CUDA_LONG N = static_cast<CUDA_LONG>(input_size);
  // int blocksPerGrid = CeilDiv(N, kNumElementsPerThread * kNumThreadsPerBlock);
  // fast_divmod block_size_including_axis_dim_div = fast_divmod(block_size_including_axis_dim);
  // fast_divmod block_size_inside_axis_dim_div = fast_divmod(block_size_inside_axis_dim);
  // fast_divmod split_size_div = fast_divmod(static_cast<int>(split_size));
  S2SModelSplitQuickGeluKernel<T><<<1, 1, 0, stream>>>(num_outputs, input_data, output_data);



  // return Status::OK();



  // TODO: Call Split Function, it will have two outputs, out1, out2

  // TODO: Call QuickGelu on second output (OP_QuickGelu/CtxQuickGelu)
  // store it as out_quickgelu

  // TODO: Multiply out1 and out_quickgelu and store it in output Y

  // T reg_Y[kElementsPerThread] = out1 * out_quickgelu;
  // *(reinterpret_cast<LoadT*>(&Y[input_idx])) = *reinterpret_cast<LoadT*>(&reg_Y[0]);


  // int num_threads_per_block = std::min<int>(static_cast<int>(CeilDiv(bias_size, kElementsPerThread)), kThreadsPerBlock);
  // const auto grid_width = CeilDiv(bias_size, kElementsPerThread * num_threads_per_block);
  // const auto grid_height = input_size / bias_size;
  // const dim3 grid_dim{static_cast<uint32_t>(grid_height), static_cast<uint32_t>(grid_width)};

  // constexpr int vec_alignment = std::alignment_of<aligned_vector<T, kElementsPerThread>>::value;

  // // Calling the Split kernel
  // S2SModelSplitQuickGeluKernel<T><<<blocksPerGrid, kNumThreadsPerBlock, 0, stream>>>(
  //   block_size_including_axis_dim_div, block_size_inside_axis_dim_div, split_size_div, num_outputs,
  //   reinterpret_cast<const ToCudaType<type>::MappedType*>(input_data), output_data, N
  // )
}

// explicit instantiations
#define SPECIALIZED_SplitQuickGelu_IMPL(T)                                                   \
  template void LaunchS2SModelSplitQuickGeluKernel<T>(hipStream_t stream, int num_outputs,  \
                                                      const T* input_data, T* output_data)

SPECIALIZED_SplitQuickGelu_IMPL(float);
SPECIALIZED_SplitQuickGelu_IMPL(half);
SPECIALIZED_SplitQuickGelu_IMPL(BFloat16);

}  // namespace cuda
}  // namespace contrib
}  // namespace onnxruntime
