#include "hip/hip_runtime.h"
// Copyright (c) Microsoft Corporation. All rights reserved.
// Licensed under the MIT License.

#include "core/providers/cuda/cu_inc/common.cuh"
#include "contrib_ops/cuda/math/s2s_split_quickgelu_fusion_impl.h"

using namespace onnxruntime::cuda;

namespace onnxruntime {
namespace contrib {
namespace cuda {

namespace {

constexpr int kElementsPerThread = GridDim::maxElementsPerThread;
// #ifdef USE_ROCM
// constexpr int kThreadsPerBlock = 512;
// #else
// constexpr int kThreadsPerBlock = GridDim::maxThreadsPerBlock;
// #endif

}  // namespace

// Need to use SplitSameSplitDimImpl (the other one works for different split sizes)

template <typename T>
__global__ void S2SModelSplitQuickGeluKernel(const int dim, const T* input, T* output) {
  // Can remove num_outputs parameter
  // CHange dim to be part of input
  // uint dim = 2;
  // printf("Output dim is %d\n", dim);
  uint input_line_stride = dim * 2;
  uint output_line_stride = dim;
  uint offset_in1 = blockIdx.x * input_line_stride + threadIdx.x*kElementsPerThread;
  // 10 el (dim = 5)
  // 0 idx, 5 idx
  //
  uint offset_in2 = offset_in1 + dim;
  uint offset_out = blockIdx.x * output_line_stride + threadIdx.x*kElementsPerThread;
  T one = static_cast<T>(1.f);
  T zero = static_cast<T>(0.f);
  // Specify alpha here or outside (is this an input )
  float alpha = 1.702f;
  T alpha_val = static_cast<T>(alpha);
  // Separate QuickGelu code in another fn
  printf("Curr kElementsPerThread %d\n", kElementsPerThread);
  printf("Curr blockIdx.x %d\n", blockIdx.x);
  printf("Curr threadIdx.x %d\n", threadIdx.x);
  printf("Curr offset_in1 %d\n", offset_in1);
  printf("Curr offset_in2 %d\n", offset_in2);
  printf("Curr offset_out %d\n", offset_out);
  // std::cout << "Curr kElementsPerThread:" << kElementsPerThread << std::endl;
  // input_size - dim
  // 5x4 (input_size = 20), dim = 2
  // 20 - 2
  // output is 5x 10 (dim = 10)
  // 5 is row number
  // 1 cuda block will process 1 row
  // kElementsPerThread
  // dim = 10K
  //
  // if threadIdx.x*kElementsPerThread < dim
  // int max_inp = 20 - dim;
  // What about this condition? (Removing if condition should improve Warp Divergence?)
  // for (uint i = 0; i < kElementsPerThread; i++) {
  for (uint i = 0; i < kElementsPerThread && threadIdx.x*kElementsPerThread + i < dim; i++){
    uint curr_in = offset_in1 + i;
    // int curr_half = curr_in / dim;
    printf("Curr Inp Outside %d\n", curr_in);
    // if (curr_half %2 == 0 && curr_in < max_inp){
    if (threadIdx.x*kElementsPerThread + i < dim) {
      printf("Curr Inp inside %d\n", curr_in);
      // std::cout << "Curr curr_in:" << curr_in << std::endl;
      T v = input[offset_in2+i] * alpha_val;
      T sigmoid = v >= zero ? one / (one + _Exp(-v)) : one - one / (one + _Exp(v));
      T quickgelu_out = input[offset_in2+i] * sigmoid;
      output[offset_out + i] = input[offset_in1 + i] * quickgelu_out;
      printf("Current output idx %d\n", offset_out + i);
      // printf("Current output value %f\n", quickgelu_out);
    }
  }

  // uint dim = 2;
  // float alpha = 1.702f;
  // uint twice_dim = 2*dim;
  // // max_dim is input size divided by 2*dim
  // uint max_dim = 5;
  // T one = static_cast<T>(1.f);
  // T zero = static_cast<T>(0.f);
  // for (uint i = 0; i < max_dim; i++){
  //   for (uint j = 0; j < dim; j++){
  //     T v = input[dim + i*twice_dim+j] * static_cast<T>(alpha);
  //     T sigmoid = v >= zero ? one / (one + _Exp(-v)) : one - one / (one + _Exp(v));
  //     T quickgelu_out = input[dim + i*twice_dim+j] * sigmoid;
  //     output[i*dim+j] = input[i*twice_dim+j] * quickgelu_out;
  //   }
  // }
  // for (uint i = 0; i < max_len / 2; i++) {
  //   T v = input[dim + i] * static_cast<T>(alpha);
  //   T one = static_cast<T>(1.f);
  //   T zero = static_cast<T>(0.f);
  //   T sigmoid = v >= zero ? one / (one + _Exp(-v)) : one - one / (one + _Exp(v));
  //   output[i] = input[i] * sigmoid;
  // }
}

// template <typename T>
// __global__ void VectorizedS2SModelSplitQuickGeluKernel(int64_t axis, const T* X, T* Y) {
//   const auto kElementsPerBlock = kElementsPerThread * blockDim.x;
//   const auto bias_idx = kElementsPerBlock * blockIdx.y + kElementsPerThread * threadIdx.x;
//   if (bias_idx >= bias_size) {
//     return;
//   }

//   const auto input_idx = bias_size * blockIdx.x + kElementsPerBlock * blockIdx.y + kElementsPerThread * threadIdx.x;

//   using LoadT = aligned_vector<T, kElementsPerThread>;

//   T reg_X[kElementsPerThread];
//   T reg_Y[kElementsPerThread];

// void LaunchS2SModelSplitQuickGeluKernel(hipStream_t stream, const size_t element_size, const int block_size_including_axis_dim,
//                                         const int block_size_inside_axis_dim, const int64_t split_size, const int num_outputs,
//                                         const void* input_data, OutputDataArray output_data, const size_t input_size) {

template <typename T>
void LaunchS2SModelSplitQuickGeluKernel(hipStream_t stream, int dim, int64_t input_size, const T* input_data, T* output_data) {
  CUDA_LONG N = static_cast<CUDA_LONG>(input_size);
  int num_threads_per_block = std::min<int>(static_cast<int>(CeilDiv(dim, kElementsPerThread)), kThreadsPerBlock);
  int num_blocks = static_cast<int>(N/(2*dim));
  printf("Final number threads per block %d\n", num_threads_per_block);
  printf("Final num blocks %d\n", num_blocks);
  S2SModelSplitQuickGeluKernel<T><<<num_blocks, num_threads_per_block, 0, stream>>>(dim, input_data, output_data);
  // S2SModelSplitQuickGeluKernel<T><<<5, 1, 0, stream>>>(dim, input_data, output_data);
  // 4x10
  // output_dim = 5
  // 1st number of blocks = 4 (number of rows = 4)
  // 2nd size of block (threadsize) = ceil(5 / kElementsPerThread)
  // 3rd is don't need dynamic size
  //
  // 4x5x10
  // 20x10
  // number of blocks = 20
  // threadsize = ceil(5/4) = 2




  // return Status::OK();



  // TODO: Call Split Function, it will have two outputs, out1, out2

  // TODO: Call QuickGelu on second output (OP_QuickGelu/CtxQuickGelu)
  // store it as out_quickgelu

  // TODO: Multiply out1 and out_quickgelu and store it in output Y

  // T reg_Y[kElementsPerThread] = out1 * out_quickgelu;
  // *(reinterpret_cast<LoadT*>(&Y[input_idx])) = *reinterpret_cast<LoadT*>(&reg_Y[0]);


  // int num_threads_per_block = std::min<int>(static_cast<int>(CeilDiv(bias_size, kElementsPerThread)), kThreadsPerBlock);
  // const auto grid_width = CeilDiv(bias_size, kElementsPerThread * num_threads_per_block);
  // const auto grid_height = input_size / bias_size;
  // const dim3 grid_dim{static_cast<uint32_t>(grid_height), static_cast<uint32_t>(grid_width)};

  // constexpr int vec_alignment = std::alignment_of<aligned_vector<T, kElementsPerThread>>::value;

  // // Calling the Split kernel
  // S2SModelSplitQuickGeluKernel<T><<<blocksPerGrid, kNumThreadsPerBlock, 0, stream>>>(
  //   block_size_including_axis_dim_div, block_size_inside_axis_dim_div, split_size_div, num_outputs,
  //   reinterpret_cast<const ToCudaType<type>::MappedType*>(input_data), output_data, N
  // )
}

// explicit instantiations
#define SPECIALIZED_SplitQuickGelu_IMPL(T)                                                   \
  template void LaunchS2SModelSplitQuickGeluKernel<T>(hipStream_t stream, int dim, int64_t input_size, \
                                                      const T* input_data, T* output_data)

SPECIALIZED_SplitQuickGelu_IMPL(float);
SPECIALIZED_SplitQuickGelu_IMPL(half);
SPECIALIZED_SplitQuickGelu_IMPL(BFloat16);

}  // namespace cuda
}  // namespace contrib
}  // namespace onnxruntime
