#include "hip/hip_runtime.h"
// Copyright (c) Microsoft Corporation. All rights reserved.
// Licensed under the MIT License.

#include "contrib_ops/cuda/sparse/sparse_attention_impl.h"
#include "contrib_ops/cuda/sparse/block_mask.h"
#include "contrib_ops/cuda/transformers/dump_cuda_tensor.h"
#include "contrib_ops/cuda/bert/rotary_embedding_impl.h"
#include "contrib_ops/cuda/bert/group_query_attention_impl.h"
#include "contrib_ops/cpu/bert/attention_common.h"
#include "contrib_ops/cuda/bert/attention_impl.h"
#include "contrib_ops/cuda/sparse/sparse_attention_v1/sparse_attention_common.h"
#include "contrib_ops/cuda/sparse/sparse_attention_v1/sparse_attention_v1_api.h"
#include "contrib_ops/cuda/sparse/sparse_attention_v2/sparse_attention_v2_api.h"
#include "contrib_ops/cuda/bert/flash_attention/flash_api.h"

namespace onnxruntime {
namespace contrib {
namespace cuda {

// Convert total_seq_len_k (total key sequence length excluding paddings) to position_ids for Prompt
__global__ void PositionIdsPrompt(const int32_t* total_seq_len_k,
                                  int64_t* position_ids,
                                  int sequence_length,
                                  int batch_size) {
  int tid = blockDim.x * blockIdx.x + threadIdx.x;
  if (tid < batch_size * sequence_length) {
    int b = tid / sequence_length;
    int s = tid % sequence_length;
    if (s < total_seq_len_k[b]) {
      position_ids[tid] = s;
    } else {
      // padding
      position_ids[tid] = 1;
    }
  }
}

// Convert total_seq_len_k (total key sequence length excluding paddings) to position_ids for Token Generation
__global__ void PositionIdsToken(const int32_t* total_seq_len_k,
                                 int64_t* position_ids,
                                 int batch_size) {
  int tid = blockDim.x * blockIdx.x + threadIdx.x;
  if (tid < batch_size) {
    position_ids[tid] = total_seq_len_k[tid] - 1;
  }
}

// Convert total_seq_len_k (total key sequence length excluding paddings) to position_ids
Status FillPositionIds(contrib::SparseAttentionParameters& parameters,
                       const int32_t* total_seq_len_k,
                       int64_t* position_ids,
                       hipStream_t stream,
                       const int max_threads_per_block) {
  const int sequence_length = parameters.sequence_length;
  const int batch_size = parameters.batch_size;
  const int bs = batch_size * sequence_length;

  int threads = max_threads_per_block;
  if (bs <= 64) {
    threads = 64;
  } else if (bs <= 128) {
    threads = 128;
  } else if (bs <= 256) {
    threads = 256;
  } else if (bs <= 512) {
    threads = 512;
  }
  const int blocks = (bs + threads - 1) / threads;

  if (parameters.sequence_length == parameters.total_sequence_length) {  // prompt
    PositionIdsPrompt<<<blocks, threads, 0, stream>>>(total_seq_len_k, position_ids, sequence_length, batch_size);
  } else {
    PositionIdsToken<<<blocks, threads, 0, stream>>>(total_seq_len_k, position_ids, batch_size);
  }

  return CUDA_CALL(hipGetLastError());
}

// Concat new key and value (BSNH format) to kv buffer (BNSH format) in place.
template <typename T>
Status LaunchConcatKVInPlace(contrib::SparseAttentionParameters& parameters,
                             SparseAttentionData<T>& data,
                             const void* new_key,
                             const void* new_value,
                             bool is_new_kv_bnsh_format,
                             hipStream_t stream,
                             const int max_threads_per_block) {
  constexpr bool is_past_kv_bnsh_format = true;
  return LaunchConcatKVInPlace(parameters.batch_size,
                               parameters.kv_num_heads,
                               parameters.head_size,
                               parameters.max_sequence_length,
                               nullptr,
                               data.seqlens_k_total,
                               parameters.sequence_length,
                               reinterpret_cast<const T*>(new_key),
                               reinterpret_cast<const T*>(new_value),
                               data.present_key,
                               data.present_value,
                               is_past_kv_bnsh_format,
                               is_new_kv_bnsh_format,
                               stream,
                               max_threads_per_block);
}

template <typename T>
Status QkvToContext_Dense(
    const hipDeviceProp_t& device_prop,
    Stream* ort_stream,
    contrib::SparseAttentionParameters& parameters,
    SparseAttentionData<T>& data) {
  assert(parameters.is_dense);

  hipStream_t stream = static_cast<hipStream_t>(ort_stream->GetHandle());
  const int max_threads_per_block = device_prop.maxThreadsPerBlock;
  const int batch_size = parameters.batch_size;
  const int sequence_length = parameters.sequence_length;
  const int num_heads = parameters.num_heads;
  const int kv_num_heads = parameters.kv_num_heads;
  const int head_size = parameters.head_size;

  const void* query;
  const void* key;
  const void* value;

  DUMP_TENSOR_INIT();

  bool q_layout = LAYOUT_BSNH;
  bool kv_layout = LAYOUT_BSNH;
  if (!parameters.is_packed_qkv) {
    query = reinterpret_cast<const void*>(data.query);
    key = reinterpret_cast<const void*>(data.key);
    value = reinterpret_cast<const void*>(data.value);
  } else {
    size_t q_size = static_cast<size_t>(batch_size * sequence_length * num_heads * head_size);
    size_t k_size = static_cast<size_t>(batch_size * sequence_length * kv_num_heads * head_size);
    auto q = reinterpret_cast<T*>(data.unpacked_qkv_buffer);
    auto k = reinterpret_cast<T*>(data.unpacked_qkv_buffer + q_size);
    auto v = reinterpret_cast<T*>(data.unpacked_qkv_buffer + q_size + k_size);

    Status status = LaunchUnpackQKV<T, LAYOUT_BSNH>(data.query, q, k, v, num_heads, kv_num_heads, head_size,
                                                    sequence_length, batch_size, stream, max_threads_per_block);
    if (status != Status::OK()) {
      return status;
    }

    query = reinterpret_cast<const void*>(q);
    key = reinterpret_cast<const void*>(k);
    value = reinterpret_cast<const void*>(v);
  }

#if DUMP_TENSOR_LEVEL > 0
  DUMP_TENSOR("dense query (BSNH)", reinterpret_cast<const T*>(query), batch_size, sequence_length, num_heads, head_size);
  DUMP_TENSOR("dense key (BSNH)", reinterpret_cast<const T*>(key), batch_size, sequence_length, kv_num_heads, head_size);
  DUMP_TENSOR("dense value (BSNH)", reinterpret_cast<const T*>(value), batch_size, sequence_length, kv_num_heads, head_size);
#endif

  if (parameters.do_rotary) {
    size_t bsh = static_cast<size_t>(parameters.batch_size * parameters.sequence_length * parameters.head_size);
    size_t q_size = bsh * static_cast<size_t>(parameters.num_heads);
    size_t k_size = bsh * static_cast<size_t>(parameters.kv_num_heads);
    auto q_buffer = reinterpret_cast<T*>(data.rotary_buffer);
    auto k_buffer = q_buffer + q_size;
    auto position_ids_buff = reinterpret_cast<int64_t*>(k_buffer + k_size);
    ORT_RETURN_IF_ERROR(FillPositionIds(parameters, data.seqlens_k_total, position_ids_buff, stream,
                                        max_threads_per_block));

    DUMP_TENSOR("position_ids", position_ids_buff, batch_size, sequence_length);

    // Launch rotary embedding kernel. This requires separated Q, K and V
    ORT_RETURN_IF_ERROR(LaunchRotaryEmbeddingKernel<T>(stream, q_buffer, reinterpret_cast<const T*>(query),
                                                       position_ids_buff, data.cos_cache, data.sin_cache,
                                                       parameters.batch_size, parameters.sequence_length,
                                                       parameters.num_heads, parameters.head_size,
                                                       parameters.rotary_dim, parameters.max_sequence_length,
                                                       /*position_ids_format*/ 1, parameters.rotary_interleaved,
                                                       max_threads_per_block, q_layout));
    ORT_RETURN_IF_ERROR(LaunchRotaryEmbeddingKernel<T>(stream, k_buffer, reinterpret_cast<const T*>(key),
                                                       position_ids_buff, data.cos_cache, data.sin_cache,
                                                       parameters.batch_size, parameters.sequence_length,
                                                       parameters.kv_num_heads, parameters.head_size,
                                                       parameters.rotary_dim, parameters.max_sequence_length,
                                                       /*position_ids_format*/ 1, parameters.rotary_interleaved,
                                                       max_threads_per_block, kv_layout));
    query = reinterpret_cast<const void*>(q_buffer);
    key = reinterpret_cast<const void*>(k_buffer);

#if DUMP_TENSOR_LEVEL > 0
    DUMP_TENSOR("query after rotary", reinterpret_cast<const T*>(query),
                batch_size, sequence_length, num_heads, head_size);
    DUMP_TENSOR("key after rotary", reinterpret_cast<const T*>(key),
                batch_size, sequence_length, kv_num_heads, head_size);
#endif
  }

  // Concat new key and value to kv buffers (in BNSH format) in place
  ORT_ENFORCE(parameters.past_present_share_buffer);
  ORT_RETURN_IF_ERROR(LaunchConcatKVInPlace(
      parameters, data, key, value, kv_layout, stream, max_threads_per_block));
  DUMP_TENSOR("key cache", data.present_key, batch_size, kv_num_heads, parameters.max_sequence_length, head_size);
  DUMP_TENSOR("value cache", data.present_value, batch_size, kv_num_heads, parameters.max_sequence_length, head_size);

  ORT_ENFORCE(q_layout == LAYOUT_BSNH);
  auto result = onnxruntime::flash::mha_fwd(
      device_prop,
      static_cast<hipStream_t>(ort_stream->GetHandle()),
      const_cast<void*>(query),
      const_cast<void*>(key),
      const_cast<void*>(value),
      data.output,
      reinterpret_cast<void*>(data.softmax_lse.get()),
      parameters.batch_size,
      parameters.num_heads,
      parameters.kv_num_heads,
      parameters.head_size,
      parameters.sequence_length,
      parameters.total_sequence_length,
      parameters.scale,
      true,  // causal
      std::is_same<T, BFloat16>::value,
      data.num_splits,
      reinterpret_cast<void*>(data.softmax_lse_accum.get()),
      reinterpret_cast<void*>(data.out_accum.get()),
      kv_layout == LAYOUT_BSNH);

// TODO: only dump to total sequence length instead of max sequence length.
#if DUMP_TENSOR_LEVEL > 0
  printf("batch_size=%d, sequence_length=%d, num_heads=%d, head_size=%d, "
         "total_sequence_length=%d, scale=%f, bsnh=%d, num_splits=%d\n",
         parameters.batch_size,
         parameters.sequence_length,
         parameters.num_heads,
         parameters.head_size,
         parameters.total_sequence_length,
         parameters.scale,
         static_cast<int>(q_layout == LAYOUT_BSNH),
         data.num_splits);
  DUMP_TENSOR("dense output (BSNH)", reinterpret_cast<const T*>(data.output), batch_size, sequence_length, num_heads, head_size);
#endif

  return result;
}

template <typename T>
Status QkvToContext_Sparse(
    const hipDeviceProp_t& device_prop,
    Stream* ort_stream,
    contrib::SparseAttentionParameters& parameters,
    SparseAttentionData<T>& data) {
  hipStream_t stream = static_cast<hipStream_t>(ort_stream->GetHandle());
  const int max_threads_per_block = device_prop.maxThreadsPerBlock;
  const int batch_size = parameters.batch_size;
  const int sequence_length = parameters.sequence_length;
  // const int present_sequence_length = parameters.max_sequence_length;
  const int num_heads = parameters.num_heads;
  const int kv_num_heads = parameters.kv_num_heads;
  const int head_size = parameters.head_size;

  const void* query;
  const void* key;
  const void* value;

  DUMP_TENSOR_INIT();

  constexpr bool q_layout = LAYOUT_BNSH;
  bool kv_layout = LAYOUT_BSNH;
  if (!parameters.is_packed_qkv) {
    static_assert(sizeof(T) == 2);
    ORT_RETURN_IF_ERROR(Transpose_BSNH_to_BNSH(
        batch_size, sequence_length, num_heads, head_size,
        reinterpret_cast<const half*>(data.query), reinterpret_cast<half*>(data.transposed_q_buffer),
        stream, max_threads_per_block));
    query = reinterpret_cast<const void*>(data.transposed_q_buffer);
    key = reinterpret_cast<const void*>(data.key);
    value = reinterpret_cast<const void*>(data.value);
  } else {
    size_t q_size = static_cast<size_t>(batch_size * sequence_length * num_heads * head_size);
    size_t k_size = static_cast<size_t>(batch_size * sequence_length * kv_num_heads * head_size);
    auto q = reinterpret_cast<T*>(data.unpacked_qkv_buffer);
    auto k = reinterpret_cast<T*>(data.unpacked_qkv_buffer + q_size);
    auto v = reinterpret_cast<T*>(data.unpacked_qkv_buffer + q_size + k_size);

    Status status = LaunchUnpackQKV<T, LAYOUT_BNSH>(data.query, q, k, v, num_heads, kv_num_heads, head_size,
                                                    sequence_length, batch_size, stream, max_threads_per_block);
    if (status != Status::OK()) {
      return status;
    }

    kv_layout = LAYOUT_BNSH;
    query = reinterpret_cast<const void*>(q);
    key = reinterpret_cast<const void*>(k);
    value = reinterpret_cast<const void*>(v);
  }

#if DUMP_TENSOR_LEVEL > 0
  DUMP_TENSOR("sparse query (BNSH)", reinterpret_cast<const T*>(query), batch_size, num_heads, sequence_length, head_size);

  if (LAYOUT_BNSH == kv_layout) {
    DUMP_TENSOR("sparse key (BNSH)", reinterpret_cast<const T*>(key), batch_size, kv_num_heads, sequence_length, head_size);
    DUMP_TENSOR("sparse value (BNSH)", reinterpret_cast<const T*>(value), batch_size, kv_num_heads, sequence_length, head_size);
  } else {
    DUMP_TENSOR("sparse key (BSNH)", reinterpret_cast<const T*>(key), batch_size, sequence_length, kv_num_heads, head_size);
    DUMP_TENSOR("sparse value (BSNH)", reinterpret_cast<const T*>(value), batch_size, sequence_length, kv_num_heads, head_size);
  }
#endif

  if (parameters.do_rotary) {
    size_t bsh = static_cast<size_t>(parameters.batch_size * parameters.sequence_length * parameters.head_size);
    size_t q_size = bsh * static_cast<size_t>(parameters.num_heads);
    size_t k_size = bsh * static_cast<size_t>(parameters.kv_num_heads);
    auto q_buffer = reinterpret_cast<T*>(data.rotary_buffer);
    auto k_buffer = q_buffer + q_size;
    auto position_ids_buff = reinterpret_cast<int64_t*>(k_buffer + k_size);
    ORT_RETURN_IF_ERROR(FillPositionIds(parameters, data.seqlens_k_total, position_ids_buff, stream,
                                        max_threads_per_block));

    DUMP_TENSOR("position_ids", position_ids_buff, batch_size, sequence_length);

    // Launch rotary embedding kernel. This requires separated Q, K and V
    ORT_RETURN_IF_ERROR(LaunchRotaryEmbeddingKernel<T>(stream, q_buffer, reinterpret_cast<const T*>(query),
                                                       position_ids_buff, data.cos_cache, data.sin_cache,
                                                       parameters.batch_size, parameters.sequence_length,
                                                       parameters.num_heads, parameters.head_size,
                                                       parameters.rotary_dim, parameters.max_sequence_length,
                                                       /*position_ids_format*/ 1, parameters.rotary_interleaved,
                                                       max_threads_per_block, q_layout));
    ORT_RETURN_IF_ERROR(LaunchRotaryEmbeddingKernel<T>(stream, k_buffer, reinterpret_cast<const T*>(key),
                                                       position_ids_buff, data.cos_cache, data.sin_cache,
                                                       parameters.batch_size, parameters.sequence_length,
                                                       parameters.kv_num_heads, parameters.head_size,
                                                       parameters.rotary_dim, parameters.max_sequence_length,
                                                       /*position_ids_format*/ 1, parameters.rotary_interleaved,
                                                       max_threads_per_block, kv_layout));
    query = reinterpret_cast<const void*>(q_buffer);
    key = reinterpret_cast<const void*>(k_buffer);

#if DUMP_TENSOR_LEVEL > 0
    DUMP_TENSOR("query after rotary", reinterpret_cast<const T*>(query),
                batch_size, num_heads, sequence_length, head_size);
    if (LAYOUT_BNSH == kv_layout) {
      DUMP_TENSOR("key after rotary", reinterpret_cast<const T*>(key),
                  batch_size, kv_num_heads, sequence_length, head_size);
    } else {
      DUMP_TENSOR("key after rotary", reinterpret_cast<const T*>(key),
                  batch_size, sequence_length, kv_num_heads, head_size);
    }
#endif
  }

  // Concat new key and value to kv buffers (in BNSH format) in place
  ORT_ENFORCE(parameters.past_present_share_buffer);
  ORT_RETURN_IF_ERROR(LaunchConcatKVInPlace(
      parameters, data, key, value, kv_layout, stream, max_threads_per_block));

  // TODO: only dump to total sequence length instead of max sequence length.
#if DUMP_TENSOR_LEVEL > 0
  DUMP_TENSOR("key cache", data.present_key, batch_size, kv_num_heads, parameters.max_sequence_length, head_size);
  DUMP_TENSOR("value cache", data.present_value, batch_size, kv_num_heads, parameters.max_sequence_length, head_size);

  DUMP_TENSOR("block_mask",
              data.kernel_layout.mask,
              data.kernel_layout.num_layout,
              data.kernel_layout.num_rows,
              data.kernel_layout.num_cols);

  DUMP_TENSOR("csr_col_indices",
              data.kernel_layout.csr_col_indices,
              data.kernel_layout.num_layout,
              data.kernel_layout.num_rows,
              data.kernel_layout.num_cols);

  DUMP_TENSOR("csr_row_indices",
              data.kernel_layout.csr_row_indices,
              data.kernel_layout.num_layout,
              data.kernel_layout.num_rows + 1);

  printf(
      "batch_size=%d, sequence_length=%d, num_heads=%d, kv_num_heads=%d head_size=%d, "
      "total_sequence_length=%d, max_sequence_length=%d scale=%f block_size=%d "
      "row_stride=%d col_stride=%d num_layout=%d\n",
      parameters.batch_size,
      parameters.sequence_length,
      parameters.num_heads,
      parameters.kv_num_heads,
      parameters.head_size,
      parameters.total_sequence_length,
      parameters.max_sequence_length,
      parameters.scale,
      data.kernel_layout.block_size,
      data.kernel_layout.num_rows + 1,
      data.kernel_layout.num_rows * data.kernel_layout.num_cols,
      data.kernel_layout.num_layout);
#endif

  int sm = device_prop.major * 10 + device_prop.minor;
  if (data.use_v2_kernel) {
    sparse_attention_v2::SparseAttentionParams params(
        ort_stream,
        sm,
        data.output,
        reinterpret_cast<const void*>(query),
        reinterpret_cast<const void*>(data.present_key),
        reinterpret_cast<const void*>(data.present_value),
        parameters.batch_size,
        parameters.sequence_length,
        parameters.num_heads,
        parameters.kv_num_heads,
        parameters.head_size,
        parameters.total_sequence_length,
        parameters.max_sequence_length,
        parameters.scale,
        data.kernel_layout.block_size,                              // kernel_block_size
        data.kernel_layout.csr_row_indices,                         // skip past_seq_len in row indices
        data.kernel_layout.csr_col_indices,                         // (num_layout, num_rows, num_cols)
        data.kernel_layout.num_rows + 1,                            // stride per head in row indices
        data.kernel_layout.num_rows * data.kernel_layout.num_cols,  // stride per head in col indices
        data.kernel_layout.num_layout,
        data.active_q_blocks,
        data.q_batch_starts,
        data.q_batch_ends,
        data.k_batch_starts,
        data.k_batch_ends,
        data.q_batch_ids,
        data.q_start_sids);

    if constexpr (std::is_same<T, BFloat16>::value) {
      ORT_RETURN_IF_ERROR(sparse_attention_v2::run_sparse_attention_bf16(params));
    } else {
      ORT_RETURN_IF_ERROR(sparse_attention_v2::run_sparse_attention_fp16(params));
    }
  } else {
    sparse_attention_v1::SparseAttentionParams params(
        ort_stream,
        sm,
        data.output,
        reinterpret_cast<const void*>(query),
        reinterpret_cast<const void*>(data.present_key),
        reinterpret_cast<const void*>(data.present_value),
        parameters.batch_size,
        parameters.sequence_length,
        parameters.num_heads,
        parameters.kv_num_heads,
        parameters.head_size,
        parameters.total_sequence_length,
        parameters.max_sequence_length,
        parameters.scale,
        data.kernel_layout.block_size,                              // kernel_block_size
        data.kernel_layout.csr_row_indices,                         // (num_layout, num_rows + 1)
        data.kernel_layout.csr_col_indices,                         // (num_layout, num_rows, num_cols)
        data.kernel_layout.num_rows + 1,                            // stride per head in row indices
        data.kernel_layout.num_rows * data.kernel_layout.num_cols,  // stride per head in col indices
        data.kernel_layout.num_layout);

    if constexpr (std::is_same<T, BFloat16>::value) {
      ORT_RETURN_IF_ERROR(sparse_attention_v1::run_sparse_attention_bf16(params));
    } else {
      ORT_RETURN_IF_ERROR(sparse_attention_v1::run_sparse_attention_fp16(params));
    }
  }

  DUMP_TENSOR("sparse output", reinterpret_cast<const T*>(data.output), batch_size, num_heads, sequence_length, head_size);

  return Status::OK();
}

template Status QkvToContext_Dense<half>(
    const hipDeviceProp_t& device_prop,
    Stream* ort_stream,
    contrib::SparseAttentionParameters& parameters,
    SparseAttentionData<half>& data);

template Status QkvToContext_Dense<BFloat16>(
    const hipDeviceProp_t& device_prop,
    Stream* ort_stream,
    contrib::SparseAttentionParameters& parameters,
    SparseAttentionData<BFloat16>& data);

template Status QkvToContext_Sparse<half>(
    const hipDeviceProp_t& device_prop,
    Stream* ort_stream,
    contrib::SparseAttentionParameters& parameters,
    SparseAttentionData<half>& data);

template Status QkvToContext_Sparse<BFloat16>(
    const hipDeviceProp_t& device_prop,
    Stream* ort_stream,
    contrib::SparseAttentionParameters& parameters,
    SparseAttentionData<BFloat16>& data);

}  // namespace cuda
}  // namespace contrib
}  // namespace onnxruntime
