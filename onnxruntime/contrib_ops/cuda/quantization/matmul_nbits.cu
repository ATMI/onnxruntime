#include "hip/hip_runtime.h"
// Modifications: scaling is moved from masked softmax to the gemm before that.
// Copyright (c) Microsoft Corporation. All rights reserved.
// Licensed under the MIT License.

#include <hipcub/hipcub.hpp>
#include <hipblas.h>
#include <hip/hip_fp16.h>
#include <hip/hip_math_constants.h>
#include "core/providers/cuda/cu_inc/common.cuh"
#include "core/providers/cuda/cuda_common.h"
#include "matmul_nbits.cuh"

using namespace onnxruntime::cuda;
using namespace hipcub;

namespace onnxruntime {
namespace contrib {
namespace cuda {

template <typename T>
__device__ __forceinline__ T WarpUniform(T value) {
  struct {
    union {
      T value;
      uint32_t asInt;
    };
  } p;
  p.value = value;
  p.asInt = __shfl_sync(0xffffffff, (unsigned)p.asInt, 0);
  return p.value;
}

#if !defined(__CUDA_ARCH__) || __CUDA_ARCH__ >= 530
// Convert 8 4bits integer stored in one uint32_t to 8 halfs.
// 8 4bits with order 0,1,2,3,4,5,6,7,8 will be converted to 8 halfs with order 0,4,1,5,2,6,3,7
__device__ __forceinline__ void Convert8xInt4To8xHalfs(uint32_t value, half2* half_2x4) {
  uint32_t* h = reinterpret_cast<uint32_t*>(half_2x4);

  // From https://github.com/NVIDIA/FasterTransformer/blob/main/src/fastertransformer/cutlass_extensions/include/cutlass_extensions/interleaved_numeric_conversion.h
  // First, we extract the i4s and construct an intermediate fp16 number.
  constexpr uint32_t kImmLut = (0xf0 & 0xcc) | 0xaa;
  constexpr uint32_t kBottomMask = 0x000f000f;
  constexpr uint32_t kTopMask = 0x00f000f0;
  constexpr uint32_t kI4sToF16sMagicNum = 0x64006400;

  // Note that the entire sequence only requires 1 shift instruction. This is thanks to the register packing
  // format and the fact that we force our integers to be unsigned, and account for this in the fp16 subtractions.
  // In addition, I exploit the fact that sub and fma have the same throughput in order to convert elt_23 and
  // elt_67 to fp16 without having to shift them to the bottom bits before hand.

  // Shift right by 8 to now consider elt_45 and elt_67. Issue first to hide RAW dependency if we issue
  // immediately before required.
  const uint32_t top_i4s = value >> 8;
  // Extract elt_01 - (i4s & 0x000f000f) | 0x64006400
  asm volatile("lop3.b32 %0, %1, %2, %3, %4;\n"
               : "=r"(h[0])
               : "r"(value), "n"(kBottomMask), "n"(kI4sToF16sMagicNum), "n"(kImmLut));
  // Extract elt_23 (i4s & 0x00f000f0) | 0x64006400
  asm volatile("lop3.b32 %0, %1, %2, %3, %4;\n"
               : "=r"(h[1])
               : "r"(value), "n"(kTopMask), "n"(kI4sToF16sMagicNum), "n"(kImmLut));
  // Extract elt_45 (top_i4s & 0x000f000f) | 0x64006400
  asm volatile("lop3.b32 %0, %1, %2, %3, %4;\n"
               : "=r"(h[2])
               : "r"(top_i4s), "n"(kBottomMask), "n"(kI4sToF16sMagicNum), "n"(kImmLut));
  // Extract elt_67 (top_i4s & 0x00f000f0) | 0x64006400
  asm volatile("lop3.b32 %0, %1, %2, %3, %4;\n"
               : "=r"(h[3])
               : "r"(top_i4s), "n"(kTopMask), "n"(kI4sToF16sMagicNum), "n"(kImmLut));

  // I use inline PTX below because I am not sure if the compiler will emit float2half instructions if I use the
  // half2 ctor. In this case, I chose performance reliability over code readability.

  // This is the half2 {1024, 1024} represented as an integer.
  constexpr uint32_t kFp16TopMagicNum = 0x64006400;
  // This is the half2 {1 / 16, 1 / 16} represented as an integer.
  constexpr uint32_t kOneSixteenth = 0x2c002c00;
  // This is the half2 {-64, -64} represented as an integer.
  constexpr uint32_t kNeg64 = 0xd400d400;

  // Finally, we construct the output numbers.
  // Convert elt_01
  asm volatile("sub.f16x2 %0, %1, %2;\n" : "=r"(h[0]) : "r"(h[0]), "r"(kFp16TopMagicNum));
  // Convert elt_23
  asm volatile("fma.rn.f16x2 %0, %1, %2, %3;\n" : "=r"(h[1]) : "r"(h[1]), "r"(kOneSixteenth), "r"(kNeg64));
  // Convert elt_45
  asm volatile("sub.f16x2 %0, %1, %2;\n" : "=r"(h[2]) : "r"(h[2]), "r"(kFp16TopMagicNum));
  // Convert elt_67
  asm volatile("fma.rn.f16x2 %0, %1, %2, %3;\n" : "=r"(h[3]) : "r"(h[3]), "r"(kOneSixteenth), "r"(kNeg64));
}

__device__ __forceinline__ float AccumulateEightElements(uint32_t values_quant, half scale, uint8_t zp, const half* a, half* sums) {
  half2 scale_half2 = {scale, scale};
  half zp_adjust = -scale * __short2half_rn(zp);
  half2 zp_adjust2 = {zp_adjust, zp_adjust};
  uint4 vec_a = *(reinterpret_cast<const uint4*>(a));

  constexpr uint32_t kLowHalf2 = 0x5410;
  constexpr uint32_t kHighHalf2 = 0x7632;

  uint4 vec_permuted;
  asm volatile("prmt.b32 %0, %1, %2, %3;\n" : "=r"(vec_permuted.x) : "r"(vec_a.x), "r"(vec_a.z), "r"(kLowHalf2));
  asm volatile("prmt.b32 %0, %1, %2, %3;\n" : "=r"(vec_permuted.y) : "r"(vec_a.x), "r"(vec_a.z), "r"(kHighHalf2));
  asm volatile("prmt.b32 %0, %1, %2, %3;\n" : "=r"(vec_permuted.z) : "r"(vec_a.y), "r"(vec_a.w), "r"(kLowHalf2));
  asm volatile("prmt.b32 %0, %1, %2, %3;\n" : "=r"(vec_permuted.w) : "r"(vec_a.y), "r"(vec_a.w), "r"(kHighHalf2));

  half2 elements[4];  // [04, 15, 26, 37]

  Convert8xInt4To8xHalfs(values_quant, elements);

  half2 v0 = elements[0] * scale_half2 + zp_adjust2;
  half2 v1 = elements[1] * scale_half2 + zp_adjust2;
  half2 v2 = elements[2] * scale_half2 + zp_adjust2;
  half2 v3 = elements[3] * scale_half2 + zp_adjust2;

  half2* sums_half2 = reinterpret_cast<half2*>(sums);
  sums_half2[0] = sums_half2[0] + v0 * (*(reinterpret_cast<half2*>(&(vec_permuted.x))));
  sums_half2[1] = sums_half2[1] + v1 * (*(reinterpret_cast<half2*>(&(vec_permuted.y))));
  sums_half2[2] = sums_half2[2] + v2 * (*(reinterpret_cast<half2*>(&(vec_permuted.z))));
  sums_half2[3] = sums_half2[3] + v3 * (*(reinterpret_cast<half2*>(&(vec_permuted.w))));
}
#else
__device__ __forceinline__ float AccumulateEightElements(uint32_t values_quant, half scale, uint8_t zp, const half* a, half* sums) {
  half2 scale_half2 = {scale, scale};
  half zp_adjust = -scale * __short2half_rn(zp);
  half2 zp_adjust2 = {zp_adjust, zp_adjust};
  uint4 vec_a = *(reinterpret_cast<const uint4*>(a));

  half2 element01 = __halves2half2(__uint2half_rn(values_quant & 0xF), __uint2half_rn((values_quant >> 4) & 0xF));
  half2 element23 = __halves2half2(__uint2half_rn((values_quant >> 8) & 0xF), __uint2half_rn((values_quant >> 12) & 0xF));
  half2 element45 = __halves2half2(__uint2half_rn((values_quant >> 16) & 0xF), __uint2half_rn((values_quant >> 20) & 0xF));
  half2 element67 = __halves2half2(__uint2half_rn((values_quant >> 24) & 0xF), __uint2half_rn((values_quant >> 28) & 0xF));

  half2 v0 = element01 * scale_half2 + zp_adjust2;
  half2 v1 = element23 * scale_half2 + zp_adjust2;
  half2 v2 = element45 * scale_half2 + zp_adjust2;
  half2 v3 = element67 * scale_half2 + zp_adjust2;

  half2* sums_half2 = reinterpret_cast<half2*>(sums);
  sums_half2[0] = sums_half2[0] + v0 * (*(reinterpret_cast<half2*>(&(vec_a.x))));
  sums_half2[1] = sums_half2[1] + v1 * (*(reinterpret_cast<half2*>(&(vec_a.y))));
  sums_half2[2] = sums_half2[2] + v2 * (*(reinterpret_cast<half2*>(&(vec_a.z))));
  sums_half2[3] = sums_half2[3] + v3 * (*(reinterpret_cast<half2*>(&(vec_a.w))));
}
#endif

__device__ __forceinline__ float AccumulateEightElements(uint32_t values_quant, float scale, uint8_t zp, const float* a, float* sums) {
  float4 a_vec_0 = *(reinterpret_cast<const float4*>(a));
  float4 a_vec_1 = *(reinterpret_cast<const float4*>(a + 4));

  float zp_adjust = -scale * zp;
  float v0 = float(values_quant & 0xF) * scale + zp_adjust;
  float v1 = float((values_quant >> 4) & 0xF) * scale + zp_adjust;
  float v2 = float((values_quant >> 8) & 0xF) * scale + zp_adjust;
  float v3 = float((values_quant >> 12) & 0xF) * scale + zp_adjust;
  float v4 = float((values_quant >> 16) & 0xF) * scale + zp_adjust;
  float v5 = float((values_quant >> 20) & 0xF) * scale + zp_adjust;
  float v6 = float((values_quant >> 24) & 0xF) * scale + zp_adjust;
  float v7 = float((values_quant >> 28) & 0xF) * scale + zp_adjust;

  sums[0] += v0 * a_vec_0.x;
  sums[1] += v1 * a_vec_0.y;
  sums[2] += v2 * a_vec_0.z;
  sums[3] += v3 * a_vec_0.w;
  sums[4] += v4 * a_vec_1.x;
  sums[5] += v5 * a_vec_1.y;
  sums[6] += v6 * a_vec_1.z;
  sums[7] += v7 * a_vec_1.w;
}

constexpr int kColsPerThreadBlock = 8;
constexpr int kWarpSize = 32;

// kernel for 4bits quantized gemv, i.e., computing A(1,K) x B(K, N)
// B(K, N) is quantized blockwise with 4bits and stored as [N, (K + block_size - 1)/block_size, blob]
// The thread block size is (kWarpSize, kColsPerThreadBlock) and grid size is (N/kColsPerThreadBlock, 1)
// Each thread block computes [1, K] x [kColsPerThreadBlock, (K + block_size - 1)/block_size, blob],
//     i.e., computing kColsPerThreadBlock per block and a warp reduce (1, K) x (K)
template <class T, int block_size, bool has_zero_point>
__global__ void __launch_bounds__(kWarpSize* kColsPerThreadBlock) MatMulFloatInt4Kernel(
    T* output,
    const T* a_data,
    const uint8_t* b_data_quant,
    const T* scales_data,
    const uint8_t* zero_points,
    int m,
    int n,
    int k,
    int blocks_per_K) {
  const int n_block_id = blockIdx.x;
  const int m_id = blockIdx.y;
  const int lane_id = threadIdx.x;
  const int warp_id = WarpUniform(threadIdx.y);
  const int n_id = n_block_id * kColsPerThreadBlock + warp_id;
  constexpr int k_per_iter = 256;

  extern __shared__ char shared_buffer[];
  // load scale to shared buffer
  T* b_scale_vec = (T*)shared_buffer;
  int offset = n_block_id * kColsPerThreadBlock * blocks_per_K;
  for (int i = warp_id * kWarpSize + lane_id; i < kColsPerThreadBlock * blocks_per_K; i += kColsPerThreadBlock * kWarpSize) {
    b_scale_vec[i] = scales_data[offset + i];
  }

  uint8_t* b_zp_vec;
  (void)b_zp_vec;
  if constexpr (has_zero_point) {
    b_zp_vec = reinterpret_cast<uint8_t*>(b_scale_vec + kColsPerThreadBlock * blocks_per_K);
    const int b_zp_k = (blocks_per_K + 1) / 2;
    int zp_offset = n_block_id * kColsPerThreadBlock * b_zp_k;
    for (int i = warp_id * kWarpSize + lane_id; i < kColsPerThreadBlock * b_zp_k; i += kColsPerThreadBlock * kWarpSize) {
      b_zp_vec[2 * i] = (zero_points[zp_offset + i] & 0x0f);
      b_zp_vec[2 * i + 1] = (zero_points[zp_offset + i] >> 4);
    }
    b_zp_vec += warp_id * b_zp_k * 2;
  }
  __syncthreads();

  a_data += m_id * k + (lane_id << 3);

  b_scale_vec += warp_id * blocks_per_K;

  T sums[8] = {0.f, 0.f, 0.f, 0.f, 0.f, 0.f, 0.f, 0.f};
  int k_id = 0;
  int t_meta_k = lane_id * 8 / block_size;
  b_data_quant += n_id * blocks_per_K * (block_size / 2) + lane_id * 4;

#define UnRollReduction(unroll_size)                                                              \
  do {                                                                                            \
    constexpr int kUnroll = unroll_size;                                                          \
    constexpr int kUnrollMask = 0xffffffff & (~(kUnroll * k_per_iter - 1));                       \
    for (; k_id < (k & kUnrollMask); k_id += kUnroll * k_per_iter) {                              \
      _Pragma("unroll") for (int i = 0; i < kUnroll; i++) {                                       \
        uint32_t value = *(reinterpret_cast<const uint32_t*>(b_data_quant + k_per_iter / 2 * i)); \
        T scale = b_scale_vec[t_meta_k + k_per_iter / block_size * i];                            \
        uint8_t zp = 8;                                                                           \
        if constexpr (has_zero_point) {                                                           \
          zp = b_zp_vec[t_meta_k + k_per_iter / block_size * i];                                  \
        }                                                                                         \
        AccumulateEightElements(value, scale, zp, a_data + k_id + i * k_per_iter, sums);          \
      }                                                                                           \
      b_data_quant += k_per_iter / 2 * kUnroll;                                                   \
      t_meta_k += k_per_iter / block_size * kUnroll;                                              \
    }                                                                                             \
  } while (false)

  UnRollReduction(16);
  UnRollReduction(4);
  UnRollReduction(1);
#undef UnRollReduction

  // handle reminder
  if (k_id + lane_id * 8 < k) {
    uint32_t value = *(reinterpret_cast<const uint32_t*>(b_data_quant));
    T scale = b_scale_vec[t_meta_k];
    uint8_t zp = 8;
    if constexpr (has_zero_point) {
      zp = b_zp_vec[t_meta_k];
    }
    AccumulateEightElements(value, scale, zp, a_data + k_id, sums);
  }

  float sum = (float)(sums[0] + sums[1] + sums[2] + sums[3] + sums[4] + sums[5] + sums[6] + sums[7]);
  // warp reduction
  for (int i = 16; i > 0; i = i / 2) {
    sum += __shfl_down_sync(0xffffffff, sum, i);
  }

  if (lane_id == 0) {
    output[m_id * n + n_id] = sum;
  }
}  // namespace cuda

template <class T>
bool TryMatMul4Bits(
    T* output,
    const T* a_data,
    const uint8_t* b_data_quant,
    const T* scales_data,
    const uint8_t* zero_points,
    int m,
    int n,
    int k,
    int block_size,
    int shared_mem_per_block,
    hipStream_t stream) {
  if (n % kColsPerThreadBlock != 0 || k % 8 != 0 || m > 1) {
    return false;
  }
  dim3 blocks((n + kColsPerThreadBlock - 1) / kColsPerThreadBlock, m);
  dim3 threads(kWarpSize, kColsPerThreadBlock);
  int blocks_per_K = (k + block_size - 1) / block_size;
  int shared_mem_size = sizeof(T) * blocks_per_K * kColsPerThreadBlock +
                        (zero_points != nullptr ? (blocks_per_K + 1) / 2 * kColsPerThreadBlock * 2 : 0);
  if (shared_mem_size > shared_mem_per_block) {
    return false;
  }

#define MatMulFloatInt4KernelDispatch(block_size)                                              \
  if (nullptr != zero_points) {                                                                \
    MatMulFloatInt4Kernel<T, block_size, true><<<blocks, threads, shared_mem_size, stream>>>(  \
        output, a_data, b_data_quant, scales_data, zero_points, m, n, k, blocks_per_K);        \
  } else {                                                                                     \
    MatMulFloatInt4Kernel<T, block_size, false><<<blocks, threads, shared_mem_size, stream>>>( \
        output, a_data, b_data_quant, scales_data, zero_points, m, n, k, blocks_per_K);        \
  }

  if (16 == block_size) {
    MatMulFloatInt4KernelDispatch(16);
  } else if (32 == block_size) {
    MatMulFloatInt4KernelDispatch(32);
  } else if (64 == block_size) {
    MatMulFloatInt4KernelDispatch(64);
  } else if (128 == block_size) {
    MatMulFloatInt4KernelDispatch(128);
  } else {
    ORT_THROW("block size ", block_size, " is not supported");
  }

#undef MatMulFloatInt4KernelDispatch

  return true;
}

template bool TryMatMul4Bits<float>(
    float* output,
    const float* a_data,
    const uint8_t* b_data_quant,
    const float* scales_data,
    const uint8_t* zero_points,
    int m,
    int n,
    int k,
    int block_size,
    int shared_mem_per_block,
    hipStream_t stream);

template bool TryMatMul4Bits<half>(
    half* output,
    const half* a_data,
    const uint8_t* b_data_quant,
    const half* scales_data,
    const uint8_t* zero_points,
    int m,
    int n,
    int k,
    int block_size,
    int shared_mem_per_block,
    hipStream_t stream);


namespace GPTQPacking {
constexpr int kBlockSize = 256;
constexpr int kNumWaves = 32;
const int width_element_per_block = 32 * 2;
template <unsigned int WarpSize>
__device__ __forceinline__ float warpReduceSum(float sum) {
  if (WarpSize >= 32)
    sum += __shfl_down_sync(0xffffffff, sum, 16);  // 0-16, 1-17, 2-18, etc.
  if (WarpSize >= 16)
    sum += __shfl_down_sync(0xffffffff, sum, 8);  // 0-8, 1-9, 2-10, etc.
  if (WarpSize >= 8)
    sum += __shfl_down_sync(0xffffffff, sum, 4);  // 0-4, 1-5, 2-6, etc.
  if (WarpSize >= 4)
    sum += __shfl_down_sync(0xffffffff, sum, 2);  // 0-2, 1-3, 4-6, 5-7, etc.
  if (WarpSize >= 2)
    sum += __shfl_down_sync(0xffffffff, sum, 1);  // 0-1, 2-3, 4-5, etc.
  return sum;
}
template <typename T>
__global__ void MatMulW4A16Kernel(T* out, const T* inA, const uint32_t* inB, const T* scales, const uint32_t* qzeros,
                          uint32_t groupsize, const uint32_t MATRIX_M, const uint32_t MATRIX_K, const uint32_t MATRIX_N) {
  const uint32_t block_k = ((MATRIX_K + 31) / 32 + 7) / 8 * 8;

  int bid = blockIdx.x;
  __shared__ float bsum[2][32][32 + 1];
  float sum[2] = {0, 0};
  int y_start = threadIdx.y * block_k;

  half2 res2 = {};
  half2 res2_1 = {};

  const half2* inA_start = (const half2*)(inA + blockIdx.y * MATRIX_K + y_start);

  int n_offset_x = bid * width_element_per_block + threadIdx.x * 2;

  int start_group_id = (y_start / groupsize);
  int compressed_idx = threadIdx.x % 4;
  half2 scale = ((const half2*)(scales + start_group_id * MATRIX_N + n_offset_x))[0];
  uint32_t qzero_p = qzeros == nullptr ? 0x88888888 : ((qzeros + n_offset_x / 8 + start_group_id * ((MATRIX_N + 7) / 8)))[0];
  half2 hzero = __halves2half2(
      __int2half_rn((qzero_p >> (8 * (compressed_idx))) & 0xf),
      __int2half_rn(((qzero_p) >> (8 * (compressed_idx) + 4)) & 0xf));
  half2 scale_h0 = __half2half2(scale.x);
  half2 scale_h1 = __half2half2(scale.y);
  half2 hzero_scale_0 = __half2half2(hzero.x * scale.x);
  half2 hzero_scale_1 = __half2half2(hzero.y * scale.y);

#pragma unroll
  for (int i = 0; i < block_k / 2; i += 4) {  // read half2 * 4
    res2 = {};
    res2_1 = {};
    int k_offset = y_start + i * 2;
    int g_id = k_offset / groupsize;

    if (g_id > start_group_id) {
      scale = ((const half2*)(scales + g_id * MATRIX_N + n_offset_x))[0];
      qzero_p = ((qzeros + n_offset_x / 8 +
                  g_id * ((MATRIX_N + 7) / 8)))[0];
      hzero = __halves2half2(
          __int2half_rn((qzero_p >> (8 * (compressed_idx))) & 0xf),
          __int2half_rn(((qzero_p) >> (8 * (compressed_idx) + 4)) & 0xf));
      scale_h0 = __half2half2(scale.x);
      scale_h1 = __half2half2(scale.y);
      hzero_scale_0 = __half2half2(hzero.x * scale.x);
      hzero_scale_1 = __half2half2(hzero.y * scale.y);
      start_group_id = g_id;
    }

    const uint32_t* hinB = inB + n_offset_x + k_offset / 8 * MATRIX_N;
    uint32_t vbInt1 =
        (n_offset_x < MATRIX_N && (k_offset < MATRIX_K)) ? hinB[0] : int32_t(0);
    uint32_t vbInt2 = (n_offset_x + 1 < MATRIX_N && (k_offset < MATRIX_K))
                          ? (hinB)[1]
                          : int32_t(0);
    half2 vb[8];
    const uint8_t* qweight_p1 = (const uint8_t*)&vbInt1;
    const uint8_t* qweight_p2 = (const uint8_t*)&vbInt2;

#pragma unroll
    for (int j = 0; j < 4; j++) {
      // vb[j] = __halves2half2(__int2half_rn(((vbInt1 >> (j * 8))) & 0xF),
      //                        __int2half_rn(((vbInt1) >> (j*8+4)) & 0xF));
      // vb[j + 4] = __halves2half2(__int2half_rn(((vbInt2)>>(j*8)) & 0xF),
      //                            __int2half_rn((((vbInt2) >> (j*8+4))) &
      //                            0xF));
      vb[j] = __halves2half2(__int2half_rn(((*(qweight_p1 + j))) & 0xF),
                             __int2half_rn(((*(qweight_p1 + j)) >> 4) & 0xF));
      vb[j + 4] =
          __halves2half2(__int2half_rn(((*(qweight_p2 + j))) & 0xF),
                         __int2half_rn((((*(qweight_p2 + j)) >> 4)) & 0xF));
    }

    half2 va[4];
    va[0] = (k_offset < MATRIX_K) ? ((inA_start))[i] : res2;
    va[1] = (k_offset + 1 < MATRIX_K) ? ((inA_start))[i + 1] : res2;
    va[2] = (k_offset + 2 < MATRIX_K) ? ((inA_start))[i + 2] : res2;
    va[3] = (k_offset + 3 < MATRIX_K) ? ((inA_start))[i + 3] : res2;

#pragma unroll
    for (int j = 0; j < 4; j++) {
      vb[j] = __hfma2(scale_h0, vb[j], -hzero_scale_0);
      res2 = __hfma2(va[j], vb[j], res2);
      vb[4 + j] = __hfma2(scale_h1, vb[4 + j], -hzero_scale_1);
      res2_1 = __hfma2(va[j], vb[4 + j], res2_1);
    }

    sum[0] += __half2float(res2.x) + __half2float(res2.y);
    sum[1] += __half2float(res2_1.x) + __half2float(res2_1.y);
  }
  // sum[0] += __half2float(res2.x);
  // sum[1] +=  __half2float(res2.y);
  bsum[0][threadIdx.x][threadIdx.y] = sum[0];
  bsum[1][threadIdx.x][threadIdx.y] = sum[1];

  __syncthreads();
  sum[0] = 0;
  sum[1] = 0;

#pragma unroll
  for (int i = 0; i < 2; i++) {
    sum[i] = bsum[i][threadIdx.y][threadIdx.x];
    __syncthreads();
    sum[i] = warpReduceSum<32>(sum[i]);
    if (threadIdx.x == 0) {
      out[+blockIdx.y * MATRIX_N + bid * width_element_per_block +
          threadIdx.y * 2 + i] = __float2half_rn(sum[i]);
    }
  }
}

constexpr int kBlockOutput = 32;
constexpr int kMaxInputBatchInThread = 1;

template <typename scalar_t, int WBITS>
__global__ void MatMulW4A16GidxKernel(const scalar_t* __restrict__ input,
                       const int* __restrict__ qweight, scalar_t* __restrict__ output,
                       const scalar_t* __restrict__ scales,
                       const int* __restrict__ qzeros,
                       const int* __restrict__ g_idx, uint32_t mat_m,
                       uint32_t mat_k, uint32_t mat_n, uint32_t zero_width) {
  const int num_thread_group = kBlockSize / kNumWaves;
  const int thread_num_k = (mat_k + num_thread_group - 1) / num_thread_group;
  const int thread_idx_group = threadIdx.y;
  const int thread_group_start = thread_idx_group * thread_num_k;

  const int compress_group_size = 32 / WBITS;
  const int max_num_in_bits = (1 << WBITS) - 1;

  const int weight_x = blockIdx.x * kBlockOutput + threadIdx.x;

  __shared__ float blocksum[kMaxInputBatchInThread][num_thread_group]
                           [kBlockOutput];
  float sum[kMaxInputBatchInThread];
#pragma unroll
  for (int bid = 0; bid < kMaxInputBatchInThread; bid++) {
    sum[bid] = 0;
  }
  const int end_k = min(mat_k, thread_group_start + thread_num_k);
  int input_start_y = blockIdx.y * kMaxInputBatchInThread;
  int input_end_y = min(mat_m, input_start_y + kMaxInputBatchInThread);
  int len_input_y = input_end_y - input_start_y;
  for (int weight_y = thread_group_start; weight_y < end_k; weight_y++) {
    scalar_t input_vec[kMaxInputBatchInThread];
    for (int bid = 0; bid < len_input_y; bid++) {
      input_vec[bid] = input[(input_start_y + bid) * mat_k + weight_y];
    }
    int scale_row = g_idx[weight_y];

    scalar_t scale_v = scales[scale_row * mat_n + weight_x];
    uint32_t zero_v =
        qzeros == nullptr
            ? 0x88888888
            : qzeros[scale_row * zero_width + (weight_x / compress_group_size)];
    int zero_ind = weight_x % compress_group_size;
    uint8_t zv1 = (zero_v >> (zero_ind * WBITS)) & max_num_in_bits;

    scalar_t scale_zeros = __hmul(scale_v, __ushort2half_rn(zv1));

    uint32_t weight_int = qweight[(weight_y / compress_group_size) * mat_n + weight_x];
    int weight_ind = (weight_y) % compress_group_size;
    uint8_t wv1 = (weight_int >> (weight_ind * WBITS)) & max_num_in_bits;
    scalar_t wv = __ushort2half_rn(wv1);
    scalar_t weight = __hfma(wv, scale_v, -scale_zeros);
    // sum = __hfma(weight, input_v, sum);
    for (int bid = 0; bid < len_input_y; bid++) {
      sum[bid] += __half2float(weight * input_vec[bid]);
    }
  }
  for (int bid = 0; bid < len_input_y; bid++) {
    if constexpr (!std::is_same<scalar_t, float>::value) {
      blocksum[bid][thread_idx_group][threadIdx.x] = sum[bid];  //__half2float(sum);
    } else {
      blocksum[bid][thread_idx_group][threadIdx.x] = sum[bid];
    }
  }
  for (unsigned int s = 1; s < num_thread_group; s *= 2) {
    __syncthreads();
    int index = 2 * s * thread_idx_group;
    if (index < num_thread_group) {
      for (int bid = 0; bid < len_input_y; bid++) {
        blocksum[bid][index][threadIdx.x] +=
            blocksum[bid][index + s][threadIdx.x];
      }
    }
  }
  for (int bid = 0; bid < len_input_y; bid++) {
    if (thread_idx_group == 0) {
      if constexpr (!std::is_same<scalar_t, float>::value) {
        output[(input_start_y + bid) * mat_n + blockIdx.x * kBlockOutput +
               threadIdx.x] = __float2half_rn(blocksum[bid][0][threadIdx.x]);
      } else {
        output[(input_start_y + bid) * mat_n + blockIdx.x * kBlockOutput +
               threadIdx.x] = blocksum[bid][0][threadIdx.x];
      }
    }
  }
}
#define FETCH_UINT2(pointer) (reinterpret_cast<const uint2*>(&(pointer))[0])
#define FETCH_HALF2(pointer) (reinterpret_cast<const half2*>(&(pointer))[0])

template <typename T, int WBITS>
__global__ void kDequantizeAndUnpackWeight248(T* out, const int32_t* qweight, const T* scale, const int32_t* qzeros,
                                              const int group_size, const int in_features, const int n) {
  int bid = blockIdx.x;
  int tid = (bid * kBlockSize + threadIdx.x);
  // const int qweight_rows = (in_features * WBITS + 31) / 32;
  const int half_n = n / 2;

  const int compress_group_size = 32 / WBITS;
  const int max_num_in_bits = (1 << WBITS) - 1;

  uint2 weight_int2 = FETCH_UINT2(qweight[tid * 2]);
  uint32_t weight_v1 = weight_int2.x;
  uint32_t weight_v2 = weight_int2.y;

  int col_ind = (tid % half_n) * 2;
  int weight_in_row = tid / half_n * compress_group_size;
  half2 scale_v = FETCH_HALF2(scale[weight_in_row / group_size * n + col_ind]);
  uint32_t zero_v = qzeros == nullptr ? 0x88888888 : qzeros[weight_in_row / group_size * (n / compress_group_size) + (col_ind) / compress_group_size];
  int zero_ind = col_ind % compress_group_size;
  uint8_t zv1 = (zero_v >> (zero_ind * WBITS)) & max_num_in_bits;
  uint8_t zv2 = (zero_v >> (zero_ind * WBITS + WBITS)) & max_num_in_bits;
  half2 scale_zeros = __hmul2(__halves2half2(__short2half_rn(zv1), __short2half_rn(zv2)), scale_v);

  half2* out_h2 = reinterpret_cast<half2*>(out);
  // decompress weights
  int remains = in_features - weight_in_row;
  if (remains >= compress_group_size) {
#pragma unroll
    for (int i = 0; i < compress_group_size; i++) {
      uint8_t wv1 = (weight_v1 >> (i * WBITS)) & max_num_in_bits;
      uint8_t wv2 = (weight_v2 >> (i * WBITS)) & max_num_in_bits;
      half2 wv = __halves2half2(__short2half_rn(wv1), __short2half_rn(wv2));
      out_h2[((weight_in_row + i) * n + col_ind) / 2] = __hfma2(wv, scale_v, -scale_zeros);
    }
  } else {
    for (int i = 0; i < remains; i++) {
      uint8_t wv1 = (weight_v1 >> (i * WBITS)) & max_num_in_bits;
      uint8_t wv2 = (weight_v2 >> (i * WBITS)) & max_num_in_bits;
      half2 wv = __halves2half2(__short2half_rn(wv1), __short2half_rn(wv2));
      out_h2[((weight_in_row + i) * n + col_ind) / 2] = __hfma2(wv, scale_v, -scale_zeros);
    }
  }
}

template <typename T, int WBITS>
__device__ __forceinline__ uchar2 IteratorQweight(const T* ptr, int idx) {
  int start_bits = idx * WBITS;
  int first = start_bits / 32;
  int end_bits = (start_bits + WBITS);
  int second = end_bits / 32;
  start_bits = start_bits % 32;
  end_bits = end_bits % 32;
  uchar2 res;
  if (first == second) {
    res.x = (ptr[first].x >> (start_bits)) & ((1 << WBITS) - 1);
    res.y = (ptr[first].y >> (start_bits)) & ((1 << WBITS) - 1);
    return res;
  } else {
    res.x = (ptr[first].x >> (start_bits));
    res.y = (ptr[first].y >> (start_bits));

    res.x |= ((ptr[second].x) & ((1 << (end_bits)) - 1)) << (32 - start_bits);
    res.y |= ((ptr[second].y) & ((1 << (end_bits)) - 1)) << (32 - start_bits);
    return res;
  }
}

template <typename T, int WBITS>
__global__ void DequantizeAndUnpackWeight3567(T* out, const uint32_t* qweight, const T* scale, const uint32_t* qzeros,
                                                 int group_size, const int in_features, const int row_n) {
  int bid = blockIdx.x;
  int tid = (bid * kBlockSize + threadIdx.x);
  const int qweight_rows = (in_features * WBITS + 31) / 32;
  __shared__ uint2 qweight_shared[WBITS * kBlockSize];
  const int half_n = row_n / 2;

  const int group_row_n = half_n * (WBITS == 6 ? 3 : WBITS);
  int total_qw = qweight_rows * half_n;

  uint2* qweight_thread = qweight_shared + WBITS * threadIdx.x;

  int qweight_start = tid / half_n * group_row_n + tid % half_n;
  const uint2* qweigh2 = (const uint2*)qweight;
#pragma unroll
  for (int j = 0; j < WBITS; j++) {
    int ind = qweight_start + half_n * j;
    qweight_thread[j] = ind < total_qw ? (qweigh2[ind]) : uint2();
  }

  const int max_num_in_bits = (1 << WBITS) - 1;
  const int col_ind = (tid % half_n);
  const int compress_group_size = 32;
  const int fp16_weight_in_row = tid / half_n * compress_group_size;
  half2 scale_v[4];
  const int scale_zero_from = fp16_weight_in_row / group_size;
  const int scale_zero_to = min(in_features / group_size - 1, (fp16_weight_in_row + compress_group_size) / group_size);

  // decompress scales
  const half2* scale2 = reinterpret_cast<const half2*>(scale);
  for (int i = 0, scale_zero_from_i = scale_zero_from; scale_zero_from_i <= scale_zero_to; scale_zero_from_i++, i++) {
    scale_v[i] = (scale2[scale_zero_from_i * half_n + col_ind]);
  }

  // decompress zeros
  uchar2 zv1[4];
  int half_col_ind = col_ind * 2;
  const int zero_col_from = half_col_ind * WBITS / 32;
  const int zero_col_to = ((half_col_ind + 1) * WBITS - 1) / 32;
  const int zero_col_to_2 = ((half_col_ind + 2) * WBITS - 1) / 32;
  const int qzero_width = (row_n * WBITS + 32 - 1) / 32;
  for (int i = 0, scale_zero_from_i = scale_zero_from; scale_zero_from_i <= scale_zero_to; scale_zero_from_i++, i++) {
    uint32_t zero_v = qzeros == nullptr ? 0x88888888 : qzeros[scale_zero_from_i * qzero_width + zero_col_from];
    const int zero_bits_last = (((half_col_ind)*WBITS) % 32);
    zv1[i].x = (zero_v >> zero_bits_last) & max_num_in_bits;
    if (zero_col_from != zero_col_to) {
      const int zero_bits_first = ((half_col_ind + 1) * WBITS) % 32;
      uint32_t zero_v1 = qzeros == nullptr ? 0x88888888 : qzeros[scale_zero_from * qzero_width + zero_col_to];
      zv1[i].x |= (zero_v1 & ((1 << zero_bits_first) - 1)) << (32 - zero_bits_last);

      zv1[i].y = (zero_v1 >> zero_bits_first) & max_num_in_bits;
    } else {
      zv1[i].y = (zero_v >> (zero_bits_last + WBITS)) & max_num_in_bits;
    }

    if (zero_col_to != zero_col_to_2) {
      const int zero_bits_first = ((half_col_ind + 2) * WBITS) % 32;
      uint32_t zero_v1 = qzeros == nullptr ? 0x88888888 : qzeros[scale_zero_from * qzero_width + zero_col_to_2];
      zv1[i].y |= (zero_v1 & ((1 << zero_bits_first) - 1)) << (32 - zero_bits_last - WBITS);
    }
  }

  half2 scale_zeros[4];
  for (int i = 0; i <= scale_zero_to - scale_zero_from; i++) {
    scale_zeros[i] = __hmul2(__halves2half2(__ushort2half_rn(zv1[i].x), __ushort2half_rn(zv1[i].y)), scale_v[i]);
  }
  half2 scale_2 = scale_v[0];
  half2 scale_zeros_2 = scale_zeros[0];

  const int out_offset = ((fp16_weight_in_row)*half_n + col_ind);
  half2* out_h2 = reinterpret_cast<half2*>(out);
  // decompress weights
  int remains = in_features - fp16_weight_in_row;
  if (remains >= compress_group_size) {
#pragma unroll
    for (int i = 0; i < compress_group_size / 2; i++) {
      uchar2 wv1 = IteratorQweight<uint2, WBITS>(qweight_thread, i);
      uchar2 wv2 = IteratorQweight<uint2, WBITS>(qweight_thread, 16 + i);

      half2 wv = __halves2half2(__ushort2half_rn(wv1.x), __ushort2half_rn(wv1.y));
      if (group_size < 32) {
        half2 scale_2 = scale_v[i / group_size];
        half2 scale_zeros_2 = scale_zeros[i / group_size];
      }
      half2 res = __hfma2(wv, scale_2, -scale_zeros_2);
      out_h2[out_offset + i * half_n] = res;

      wv = __halves2half2(__ushort2half_rn(wv2.x), __ushort2half_rn(wv2.y));
      if (group_size < 32) {
        half2 scale_2 = scale_v[(i + 16) / group_size];
        half2 scale_zeros_2 = scale_zeros[(i + 16) / group_size];
      }
      res = __hfma2(wv, scale_2, -scale_zeros_2);
      out_h2[(out_offset + (i + 16) * half_n)] = res;
    }
  } else {
    // decompress weights
    for (int i = 0; i < remains; i++) {
      uchar2 wv1 = IteratorQweight<uint2, WBITS>(qweight_thread, i);

      half2 wv = __halves2half2(__ushort2half_rn(wv1.x), __ushort2half_rn(wv1.y));
      if (group_size < 32) {
        scale_2 = scale_v[i / group_size];
        scale_zeros_2 = scale_zeros[i / group_size];
      }
      half2 res = __hfma2(wv, scale_2, -scale_zeros_2);
      out_h2[out_offset + i * half_n] = res;
    }
  }
}

template <typename scalar_t, int WBITS>
__global__ void DequantizeAndUnpackWeight357Gidx(
    scalar_t* out, const uint32_t* qweight, const scalar_t* scale, const uint32_t* qzeros,
    const int32_t* g_idx, int group_size, const int in_features, const int n,
    uint8_t add_zero_bias) {
  int bid = blockIdx.x;
  int tid = (bid * kBlockSize + threadIdx.x);
  int out_x = tid % n;
  int out_y = tid / n;
  int scale_row = g_idx[out_y];

  const int max_num_in_bits = (1 << WBITS) - 1;

  const int qzero_width = (n * WBITS + 32 - 1) / 32;
  scalar_t scale_v = scale[scale_row * n + out_x];
  uint32_t zero_v1 = 0x88888888;
  uint8_t zv1 = 0;
  if (qzeros != nullptr) {
    int start_bits = out_x * WBITS;
    int first = start_bits / 32;
    int end_bits = (start_bits + WBITS);
    int second = end_bits / 32;
    start_bits = start_bits % 32;
    end_bits = end_bits % 32;

    zero_v1 = qzeros[scale_row * qzero_width + first];
    zv1 = (zero_v1 >> start_bits) & max_num_in_bits;
    if (first != second) {
      zero_v1 = qzeros[scale_row * qzero_width + second];
      zv1 |= (zero_v1 & ((1 << end_bits) - 1)) << (32 - start_bits);
    }
  }

  scalar_t scale_zeros = __hmul(scale_v, __ushort2half_rn(zv1 + add_zero_bias));

  uint32_t weight_int = 0;
  uint8_t wv1 = 0;
  {
    int start_bits = out_y * WBITS;
    int first = start_bits / 32;
    int end_bits = (start_bits + WBITS);
    int second = end_bits / 32;
    start_bits = start_bits % 32;
    end_bits = end_bits % 32;

    weight_int = qweight[first * n + out_x];
    wv1 = (weight_int >> start_bits) & max_num_in_bits;
    if (first != second) {
      weight_int = qweight[second * n + out_x];
      wv1 |= (weight_int & ((1 << end_bits) - 1)) << (32 - start_bits);
    }
  }

  scalar_t wv = __ushort2half_rn(wv1);
  out[tid] = __hfma(wv, scale_v, -scale_zeros);
}

template <typename scalar_t, int WBITS>
__global__ void DequantizeAndUnpackWeight248Gidx(scalar_t* out, const uint32_t* qweight, const scalar_t* scale, const uint32_t* qzeros, const int32_t* g_idx,
                                               int group_size, const int in_features, const int n, uint8_t add_zero_bias) {
  int bid = blockIdx.x;
  int tid = (bid * kBlockSize + threadIdx.x);
  int out_x = tid % n;
  int out_y = tid / n;
  int scale_row = g_idx[out_y];

  const int compress_group_size = 32 / WBITS;
  const int max_num_in_bits = (1 << WBITS) - 1;

  scalar_t scale_v = scale[scale_row * n + out_x];
  uint32_t zero_v = qzeros == nullptr
                        ? 0x88888888
                        : qzeros[scale_row * (n / compress_group_size) +
                                 (out_x / compress_group_size)];
  int zero_ind = out_x % compress_group_size;
  uint8_t zv1 = (zero_v >> (zero_ind * WBITS)) & max_num_in_bits;

  scalar_t scale_zeros = __hmul(scale_v, __ushort2half_rn(zv1 + add_zero_bias));

  int weight_int = qweight[(out_y / compress_group_size) * n + out_x];
  int weight_ind = (out_y) % compress_group_size;
  uint8_t wv1 = (weight_int >> (weight_ind * WBITS)) & max_num_in_bits;
  scalar_t wv = __ushort2half_rn(wv1);
  out[tid] = __hfma(wv, scale_v, -scale_zeros);
}

void DequantWeightNbitGidx(hipStream_t stream,
                         const int32_t* qweight_i32_i, const void* scale_fp16,
                         const int32_t* qzeros_i32_i, const int32_t* g_dix,
                         void* b_fp16,
                         uint32_t mat_k, uint32_t mat_n, int bits,
                         int groupsize) {
  using scalar_t = half;
  int add_zero_bias = 0;
  dim3 gridDim = {mat_k * mat_n / kBlockSize};
  dim3 blockDim = {kBlockSize};

  const uint32_t* qweight_i32 = reinterpret_cast<const uint32_t*>(qweight_i32_i);
  const uint32_t* qzeros_i32 = reinterpret_cast<const uint32_t*>(qzeros_i32_i);
#define CASE_EVEN(WBITS)                                                                                                       \
  case WBITS:                                                                                                                  \
    DequantizeAndUnpackWeight248Gidx<scalar_t, WBITS>                                                                            \
        <<<gridDim, blockDim, 0, stream>>>(                                                                                    \
            (scalar_t*)b_fp16, qweight_i32, (scalar_t*)scale_fp16, qzeros_i32, g_dix, groupsize, mat_k, mat_n, add_zero_bias); \
    break;
#define CASE_ODD(WBITS)                                                                                                        \
  case WBITS:                                                                                                                  \
    DequantizeAndUnpackWeight357Gidx<scalar_t, WBITS>                                                                            \
        <<<gridDim, blockDim, 0, stream>>>(                                                                                    \
            (scalar_t*)b_fp16, qweight_i32, (scalar_t*)scale_fp16, qzeros_i32, g_dix, groupsize, mat_k, mat_n, add_zero_bias); \
    break;
  switch (bits) {
    CASE_EVEN(2);
    CASE_EVEN(4);
    CASE_EVEN(8);
    CASE_ODD(3);
    CASE_ODD(5);
    CASE_ODD(6);
    CASE_ODD(7);
    default:
      printf("error bits\n");
      assert(false);
  }
#undef CASE_EVEN
#undef CASE_ODD
}

void DequantWeightNbit(
    hipStream_t stream,
    const int32_t* qweight_i32,
    const void* scales_data,
    const int32_t* zeros_data,
    void* weight_out,
    uint32_t MATRIX_K,
    uint32_t MATRIX_N,
    uint32_t bits,
    uint32_t groupsize) {
  uint32_t compress_ratio = 32 / bits;
  if (bits != 2 && bits != 4 && bits != 8) {
    compress_ratio = 32;
  }
  dim3 gridDim = {(MATRIX_N / 2 * ((MATRIX_K + compress_ratio - 1) / compress_ratio) + kBlockSize - 1) / kBlockSize};
  dim3 blockDim = {kBlockSize};
#define CASE_EVEN(WBITS)                                                          \
  case WBITS:                                                                     \
    kDequantizeAndUnpackWeight248<half, WBITS><<<gridDim, blockDim, 0, stream>>>( \
        (half*)weight_out, qweight_i32, (half*)scales_data, zeros_data,           \
        groupsize, MATRIX_K, MATRIX_N);                                           \
    break;
#define CASE_ODD(WBITS)                                                           \
  case WBITS:                                                                     \
    DequantizeAndUnpackWeight3567<half, WBITS><<<gridDim, blockDim, 0, stream>>>( \
        (half*)weight_out, (const uint32_t*)qweight_i32, (half*)scales_data,      \
        (const uint32_t*)zeros_data, groupsize, MATRIX_K, MATRIX_N);              \
    break;
  switch (bits) {
    CASE_EVEN(2);
    CASE_EVEN(4);
    CASE_EVEN(8);
    CASE_ODD(3);
    CASE_ODD(5);
    CASE_ODD(6);
    CASE_ODD(7);
    default:
      break;
  }
#undef CASE_EVEN
#undef CASE_ODD
}

void TryMatMul4Bits(
    hipStream_t stream,
    const void* vec_data,
    const int32_t* mat_data,
    void* mul_out_data,
    const void* scales_data,
    const int32_t* zeros_data,
    uint32_t MATRIX_M,
    uint32_t MATRIX_K,
    uint32_t MATRIX_N,
    uint32_t groupsize) {
  const int block_k = ((MATRIX_K + 31) / 32 + 7) / 8 * 8;

  dim3 gridDim = {(MATRIX_N + width_element_per_block - 1) / width_element_per_block, MATRIX_M};
  dim3 blockDim = {32, (MATRIX_K + block_k - 1) / block_k};
  MatMulW4A16Kernel<half><<<gridDim, blockDim, 0, stream>>>(
      static_cast<half*>(mul_out_data), static_cast<const half*>(vec_data),
      reinterpret_cast<const uint32_t*>(mat_data), static_cast<const half*>(scales_data),
      reinterpret_cast<const uint32_t*>(zeros_data), groupsize, MATRIX_M, MATRIX_K, MATRIX_N);
}

template <typename T>
__forceinline__ T ceil_div(T a, T b) {
  return (a + b - 1) / b;
}

void TryMatMul4BitsGidx(
    hipStream_t stream,
    const void* input,
    const int32_t* qweight,
    void* output,
    const void* scales,
    const int32_t* qzeros,
    const int32_t* g_idx,
    const int64_t* shapes) {
  auto matricx_m = static_cast<uint32_t>(shapes[0]);
  auto matricx_k = static_cast<uint32_t>(shapes[1]);
  auto matricx_n = static_cast<uint32_t>(shapes[2]);
  auto zero_width = static_cast<uint32_t>(shapes[3]);

  dim3 blocks(ceil_div<uint32_t>(matricx_n, kBlockOutput),
              ceil_div<uint32_t>(matricx_m, kMaxInputBatchInThread));
  dim3 threads(kBlockOutput, kBlockSize / kBlockOutput);

  using scalar_t = __half;
  MatMulW4A16GidxKernel<scalar_t, 4><<<blocks, threads, 0, stream>>>(
      reinterpret_cast<const scalar_t*>(input), qweight, reinterpret_cast<scalar_t*>(output),
      reinterpret_cast<const scalar_t*>(scales), qzeros, g_idx, matricx_m, matricx_k, matricx_n, zero_width);
}
}
}  // namespace cuda
}  // namespace contrib
}  // namespace onnxruntime
