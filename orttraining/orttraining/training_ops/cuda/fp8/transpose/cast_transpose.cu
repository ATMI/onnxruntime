#include "hip/hip_runtime.h"
/*************************************************************************
 * Copyright (c) 2022-2023, NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 *
 * See LICENSE for license information.
 ************************************************************************/

#include <hip/hip_runtime.h>
#include <iostream>
#include <cfloat>

#include "orttraining/training_ops/cuda/fp8/common.h"
#include "orttraining/training_ops/cuda/fp8/utils.cuh"

namespace onnxruntime {
namespace cuda {
namespace fp8 {

template <bool full_tile, int nvec_in, int nvec_out, typename IVec, typename OVec, typename CType>
inline __device__ void cast_and_transpose_regs(const IVec (&in)[nvec_out], OVec (&out_trans)[nvec_in],
                                               typename OVec::type* output_cast_tile, const size_t current_place,
                                               const size_t stride,
                                               CType& max,  // NOLINT(*)
                                               const CType scale, const bool valid_store) {
  using T = typename OVec::type;
  using OVecC = Vec<T, nvec_in>;
#pragma unroll
  for (unsigned int i = 0; i < nvec_out; ++i) {
    OVecC out_cast;
#pragma unroll
    for (unsigned int j = 0; j < nvec_in; ++j) {
      const CType tmp = static_cast<CType>(in[i].data.elt[j]);
      const T elt_o = T(scale * tmp);

      out_cast.data.elt[j] = elt_o;
      out_trans[j].data.elt[i] = elt_o;  // thread tile transpose

      __builtin_assume(max >= 0);
      max = fmaxf(fabsf(tmp), max);
    }
    if (full_tile || valid_store) {
      out_cast.store_to(output_cast_tile, current_place + stride * i);
    }
  }
}

// STUFF TO TUNE
constexpr unsigned int n_warps_per_tile = 4;

constexpr unsigned int max_threads_per_block = 256;
static_assert(n_warps_per_tile * THREADS_PER_WARP <= max_threads_per_block);
constexpr unsigned int cast_transpose_num_threads = n_warps_per_tile * THREADS_PER_WARP;

template <int nvec_in, int nvec_out, typename CType, typename IType, typename OType>
__global__ void __launch_bounds__(cast_transpose_num_threads)
    cast_transpose_kernel(const IType* const input, OType* const output_c, OType* const output_t,
                          const CType* const scale_ptr, CType* const amax, const size_t row_length,
                          const size_t num_rows, const size_t num_tiles) {
  using IVec = Vec<IType, nvec_in>;
  using OVec = Vec<OType, nvec_out>;

  extern __shared__ char scratch[];

  const int warp_id = threadIdx.x / THREADS_PER_WARP;
  const int my_id_in_warp = threadIdx.x % THREADS_PER_WARP;
  const size_t num_tiles_x = row_length / (nvec_in * THREADS_PER_WARP);
  const size_t tile_id = blockIdx.x * blockDim.x / (THREADS_PER_WARP * n_warps_per_tile) + warp_id / n_warps_per_tile;
  if (tile_id >= num_tiles) return;
  const size_t tile_id_x = tile_id % num_tiles_x;
  const size_t tile_id_y = tile_id / num_tiles_x;

  const IType* const my_input_tile =
      input + (tile_id_x * nvec_in + tile_id_y * row_length * nvec_out) * THREADS_PER_WARP;
  OType* const my_output_c_tile =
      output_c + (tile_id_x * nvec_in + tile_id_y * row_length * nvec_out) * THREADS_PER_WARP;
  OType* const my_output_t_tile = output_t + (tile_id_y * nvec_out + tile_id_x * num_rows * nvec_in) * THREADS_PER_WARP;
  OVec* const my_scratch = reinterpret_cast<OVec*>(scratch) +
                           (my_id_in_warp + warp_id / n_warps_per_tile * THREADS_PER_WARP) * (THREADS_PER_WARP + 1);

  IVec in[2][nvec_out];
  const unsigned int warp_id_in_tile = warp_id % n_warps_per_tile;
  constexpr unsigned int n_iterations = THREADS_PER_WARP / n_warps_per_tile;
  OVec out_space[n_iterations][nvec_in];

  const size_t stride = row_length / nvec_in;
  const size_t output_stride = num_rows / nvec_out;
  size_t current_stride = warp_id_in_tile * n_iterations * nvec_out * stride;
  unsigned int my_place = (my_id_in_warp + THREADS_PER_WARP - warp_id_in_tile * n_iterations) % THREADS_PER_WARP;
  CType max = 0;
  const CType scale = scale_ptr != nullptr ? *scale_ptr : 1;
#pragma unroll
  for (unsigned int i = 0; i < nvec_out; ++i) {
    in[0][i].load_from(my_input_tile, current_stride + my_place + stride * i);
  }
#pragma unroll
  for (unsigned int i = 0; i < n_iterations; ++i) {
    const size_t current_place = current_stride + my_place;
    const unsigned int my_place_in = (my_place + THREADS_PER_WARP - 1) % THREADS_PER_WARP;
    const unsigned int current_in = (i + 1) % 2;
    if (i < n_iterations - 1) {
#pragma unroll
      for (unsigned int j = 0; j < nvec_out; ++j) {
        in[current_in][j].load_from(my_input_tile, current_stride + my_place_in + stride * (nvec_out + j));
      }
    }
    OVec out_trans[nvec_in];  // NOLINT(*)
    cast_and_transpose_regs<true>(in[current_in ^ 1], out_trans, my_output_c_tile, current_place, stride, max, scale,
                                  true);
#pragma unroll
    for (unsigned int j = 0; j < nvec_in; ++j) {
      out_space[i][j].data.vec = out_trans[j].data.vec;
    }
    my_place = (my_place + THREADS_PER_WARP - 1) % THREADS_PER_WARP;
    current_stride += nvec_out * stride;
  }

  for (unsigned int i = 0; i < nvec_in; ++i) {
#pragma unroll
    for (unsigned int j = 0; j < n_iterations; ++j) {
      my_scratch[(my_id_in_warp + THREADS_PER_WARP - j - warp_id_in_tile * n_iterations) % THREADS_PER_WARP] =
          out_space[j][i];
    }
    __syncthreads();
    my_place = (my_id_in_warp + THREADS_PER_WARP - warp_id_in_tile * n_iterations) % THREADS_PER_WARP;
    current_stride = i * output_stride + warp_id_in_tile * n_iterations * output_stride * nvec_in;
    for (unsigned int j = 0; j < n_iterations; ++j) {
      my_scratch[j + warp_id_in_tile * n_iterations].store_to(my_output_t_tile, current_stride + my_place);
      my_place = (my_place + THREADS_PER_WARP - 1) % THREADS_PER_WARP;
      current_stride += output_stride * nvec_in;
    }
    __syncthreads();
  }

  /* warp tile amax reduce*/
  max = reduce_max<cast_transpose_num_threads / THREADS_PER_WARP>(max, warp_id);

  if (threadIdx.x == 0) {
    static_assert(std::is_same<CType, float>::value);
    if (amax != nullptr) atomicMaxFloat(amax, max);
  }
}

template <int nvec_in, int nvec_out, typename CType, typename IType, typename OType>
__global__ void __launch_bounds__(cast_transpose_num_threads)
    cast_transpose_kernel_notaligned(const IType* const input, OType* const output_c, OType* const output_t,
                                     const CType* const scale_ptr, CType* const amax, const size_t row_length,
                                     const size_t num_rows, const size_t num_tiles) {
  using IVec = Vec<IType, nvec_in>;
  using OVec = Vec<OType, nvec_out>;

  extern __shared__ char scratch[];

  const int warp_id = threadIdx.x / THREADS_PER_WARP;
  const int my_id_in_warp = threadIdx.x % THREADS_PER_WARP;
  const size_t num_tiles_x = (row_length + nvec_in * THREADS_PER_WARP - 1) / (nvec_in * THREADS_PER_WARP);
  const size_t tile_id = blockIdx.x * blockDim.x / (THREADS_PER_WARP * n_warps_per_tile) + warp_id / n_warps_per_tile;
  if (tile_id >= num_tiles) return;
  const size_t tile_id_x = tile_id % num_tiles_x;
  const size_t tile_id_y = tile_id / num_tiles_x;

  const IType* const my_input_tile =
      input + (tile_id_x * nvec_in + tile_id_y * row_length * nvec_out) * THREADS_PER_WARP;
  OType* const my_output_c_tile =
      output_c + (tile_id_x * nvec_in + tile_id_y * row_length * nvec_out) * THREADS_PER_WARP;
  OType* const my_output_t_tile = output_t + (tile_id_y * nvec_out + tile_id_x * num_rows * nvec_in) * THREADS_PER_WARP;
  const size_t stride = row_length / nvec_in;
  const size_t output_stride = num_rows / nvec_out;
  const size_t row_length_rest = stride - tile_id_x * THREADS_PER_WARP;
  const size_t row_height_rest = output_stride - tile_id_y * THREADS_PER_WARP;
  const unsigned int tile_length = row_length_rest > THREADS_PER_WARP ? THREADS_PER_WARP : row_length_rest;
  const unsigned int tile_height = row_height_rest > THREADS_PER_WARP ? THREADS_PER_WARP : row_height_rest;

  OVec* const my_scratch = reinterpret_cast<OVec*>(scratch) +
                           (my_id_in_warp + warp_id / n_warps_per_tile * THREADS_PER_WARP) * (THREADS_PER_WARP + 1);

  IVec in[2][nvec_out];
  const unsigned int warp_id_in_tile = warp_id % n_warps_per_tile;
  constexpr unsigned int n_iterations = THREADS_PER_WARP / n_warps_per_tile;
  OVec out_space[n_iterations][nvec_in];

  size_t current_stride = warp_id_in_tile * n_iterations * nvec_out * stride;
  unsigned int my_place = (my_id_in_warp + THREADS_PER_WARP - warp_id_in_tile * n_iterations) % THREADS_PER_WARP;
  CType max = 0;
  const CType scale = scale_ptr != nullptr ? *scale_ptr : 1;
  {
    const bool valid_load = my_place < tile_length && warp_id_in_tile * n_iterations < tile_height;
#pragma unroll
    for (unsigned int i = 0; i < nvec_out; ++i) {
      if (valid_load) {
        in[0][i].load_from(my_input_tile, current_stride + my_place + stride * i);
      } else {
        in[0][i].clear();
      }
    }
  }
#pragma unroll
  for (unsigned int i = 0; i < n_iterations; ++i) {
    const size_t current_place = current_stride + my_place;
    const unsigned int my_place_in = (my_place + THREADS_PER_WARP - 1) % THREADS_PER_WARP;
    const unsigned int current_in = (i + 1) % 2;
    if (i < n_iterations - 1) {
      const bool valid_load = my_place_in < tile_length && warp_id_in_tile * n_iterations + i + 1 < tile_height;
#pragma unroll
      for (unsigned int j = 0; j < nvec_out; ++j) {
        if (valid_load) {
          in[current_in][j].load_from(my_input_tile, current_stride + my_place_in + stride * (nvec_out + j));
        } else {
          in[current_in][j].clear();
        }
      }
    }
    OVec out_trans[nvec_in];  // NOLINT(*)
    const bool valid_store = my_place < tile_length && warp_id_in_tile * n_iterations + i < tile_height;
    cast_and_transpose_regs<false>(in[current_in ^ 1], out_trans, my_output_c_tile, current_place, stride, max, scale,
                                   valid_store);
#pragma unroll
    for (unsigned int j = 0; j < nvec_in; ++j) {
      out_space[i][j].data.vec = out_trans[j].data.vec;
    }
    my_place = (my_place + THREADS_PER_WARP - 1) % THREADS_PER_WARP;
    current_stride += nvec_out * stride;
  }

  for (unsigned int i = 0; i < nvec_in; ++i) {
#pragma unroll
    for (unsigned int j = 0; j < n_iterations; ++j) {
      my_scratch[(my_id_in_warp + THREADS_PER_WARP - j - warp_id_in_tile * n_iterations) % THREADS_PER_WARP] =
          out_space[j][i];
    }
    __syncthreads();
    my_place = (my_id_in_warp + THREADS_PER_WARP - warp_id_in_tile * n_iterations) % THREADS_PER_WARP;
    current_stride = i * output_stride + warp_id_in_tile * n_iterations * output_stride * nvec_in;
    for (unsigned int j = 0; warp_id_in_tile * n_iterations + j < tile_length; ++j) {
      const bool valid_store = my_place < tile_height;
      if (valid_store) {
        my_scratch[j + warp_id_in_tile * n_iterations].store_to(my_output_t_tile, current_stride + my_place);
      }
      my_place = (my_place + THREADS_PER_WARP - 1) % THREADS_PER_WARP;
      current_stride += output_stride * nvec_in;
    }
    __syncthreads();
  }

  /* warp tile amax reduce*/
  max = reduce_max<cast_transpose_num_threads / THREADS_PER_WARP>(max, warp_id);

  if (threadIdx.x == 0) {
    static_assert(std::is_same<CType, float>::value);
    if (amax != nullptr) atomicMaxFloat(amax, max);
  }
}

template <typename InputType, typename OutputType>
void CastTranspose(hipStream_t stream, const InputType* input_data, OutputType* cast_output_data,
                   OutputType* transposed_output_data, const fp32* scale, fp32* amax, const size_t row_length,
                   const size_t num_rows) {
  typedef typename MappedType<InputType>::CudaType CudaInputType;
  typedef typename MappedType<OutputType>::CudaType CudaOutputType;
  const CudaInputType* cuda_input_data = reinterpret_cast<const CudaInputType*>(input_data);
  CudaOutputType* cuda_cast_output_data = reinterpret_cast<CudaOutputType*>(cast_output_data);
  CudaOutputType* cuda_transposed_output_data = reinterpret_cast<CudaOutputType*>(transposed_output_data);

// Launch specific cast-transpose kernel
#define LAUNCH_KERNEL(kernel, nvec_in, nvec_out, n_tiles, n_blocks, CudaInputType, CudaOutputType)                 \
  do {                                                                                                             \
    hipFuncSetAttribute(reinterpret_cast<const void*>(kernel<nvec_in), nvec_out, fp32, CudaInputType, CudaOutputType>,                           \
                         hipFuncAttributePreferredSharedMemoryCarveout, 100);                                     \
    kernel<nvec_in, nvec_out, fp32, CudaInputType, CudaOutputType>                                                 \
        <<<n_blocks, cast_transpose_num_threads,                                                                   \
           cast_transpose_num_threads / n_warps_per_tile*(THREADS_PER_WARP + 1) *                                  \
               sizeof(Vec<CudaOutputType, nvec_out>),                                                              \
           stream>>>(cuda_input_data, cuda_cast_output_data, cuda_transposed_output_data, scale, amax, row_length, \
                     num_rows, n_tiles);                                                                           \
  } while (false)

// Launch cast-transpose kernel for given vector sizes
#define LAUNCH_KERNEL_VEC_SIZES(load_size, store_size, CudaInputType, CudaOutputType)                            \
  do {                                                                                                           \
    constexpr int nvec_in = load_size / sizeof(CudaInputType);                                                   \
    constexpr int nvec_out = store_size / sizeof(CudaOutputType);                                                \
    const size_t n_tiles = get_n_tiles(load_size, store_size);                                                   \
    const size_t n_blocks = get_n_blocks(n_tiles);                                                               \
                                                                                                                 \
    const bool full_tile =                                                                                       \
        row_length % (nvec_in * THREADS_PER_WARP) == 0 && num_rows % (nvec_out * THREADS_PER_WARP) == 0;         \
                                                                                                                 \
    if (full_tile) {                                                                                             \
      LAUNCH_KERNEL(cast_transpose_kernel, nvec_in, nvec_out, n_tiles, n_blocks, CudaInputType, CudaOutputType); \
    } else {                                                                                                     \
      LAUNCH_KERNEL(cast_transpose_kernel_notaligned, nvec_in, nvec_out, n_tiles, n_blocks, CudaInputType,       \
                    CudaOutputType);                                                                             \
    }                                                                                                            \
  } while (false)

  // Estimate number of SMs
  // Note: H100 has 132 SMs, A100 has 108 SMs.
  // Note: Directly querying number of SMs with hipGetDeviceProperties is
  // slow (>1 ms). Consider querying once and caching.
  const int n_sms = 128;

  // Helper functions to get kernel configuration
  auto get_n_tiles = [=](size_t load_size, size_t store_size) -> int {
    constexpr size_t threads_per_warp = static_cast<size_t>(THREADS_PER_WARP);
    size_t nvec_in = load_size / sizeof(CudaInputType);
    size_t nvec_out = store_size / sizeof(CudaOutputType);
    size_t n_tiles = DIVUP(row_length, nvec_in * threads_per_warp) * DIVUP(num_rows, nvec_out * threads_per_warp);
    return n_tiles;
  };
  auto get_n_blocks = [=](size_t n_tiles) -> int {
    size_t n_warps_per_block = cast_transpose_num_threads / THREADS_PER_WARP;
    size_t n_blocks = DIVUP(n_tiles * n_warps_per_tile, n_warps_per_block);
    return n_blocks;
  };

  // Estimate optimal vector sizes and run
  // Note: Consider reducing to 2B or 1B loads/stores for
  // sufficiently small matrices. Need to consider whether reduced
  // cache efficiency is worth increased SM utilization. Also need
  // to keep in mind whether datatype can fit.
  const size_t estimated_n_tiles = get_n_tiles(8, 8);
  const size_t estimated_n_blocks = get_n_blocks(estimated_n_tiles);
  if (estimated_n_blocks >= n_sms) {
    LAUNCH_KERNEL_VEC_SIZES(8, 8, CudaInputType, CudaOutputType);
  } else {
    LAUNCH_KERNEL_VEC_SIZES(4, 4, CudaInputType, CudaOutputType);
  }

#undef LAUNCH_KERNEL
#undef LAUNCH_KERNEL_VEC_SIZES
}

#define SPECIALIZED_CAST_TRANSPOSE_IMPL(InputType, OutputType)                                                         \
  template void CastTranspose<InputType, OutputType>(hipStream_t stream, const InputType* input_data,                 \
                                                     OutputType* cast_output_data, OutputType* transposed_output_data, \
                                                     const fp32* scale, fp32* amax, const size_t row_length,           \
                                                     const size_t num_rows);

SPECIALIZED_CAST_TRANSPOSE_IMPL(MLFloat16, Float8E4M3FN)
SPECIALIZED_CAST_TRANSPOSE_IMPL(MLFloat16, Float8E5M2)

#undef SPECIALIZED_CAST_TRANSPOSE_IMPL

}  // namespace fp8
}  // namespace cuda
}  // namespace onnxruntime
